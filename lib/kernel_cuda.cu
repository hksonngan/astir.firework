#include "hip/hip_runtime.h"
#include "kernel_cuda.h"
#include <stdio.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <hip/hip_math_constants.h>

// textures
texture<float, 1, hipReadModeElementType> tex1;
texture<float, 1, hipReadModeElementType> tex_im;
texture<float, 1, hipReadModeElementType> tex_mumap;
texture<unsigned short, 1, hipReadModeElementType> tex_x1;
texture<unsigned short, 1, hipReadModeElementType> tex_y1;
texture<unsigned short, 1, hipReadModeElementType> tex_z1;
texture<unsigned short, 1, hipReadModeElementType> tex_x2;
texture<unsigned short, 1, hipReadModeElementType> tex_y2;
texture<unsigned short, 1, hipReadModeElementType> tex_z2;

/*
// DEV draw one pixel per thread, if the thread is alon the line. Too slow...
__global__ void dev_draw(float* d_im, unsigned short int* d_x1, unsigned short int* d_y1,
						 unsigned short int* d_z1, unsigned short int* d_x2, unsigned short int* d_y2,
						 unsigned short int* d_z2, int wim, int nx1, int nim) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x1, y1, z1, x2, y2, z2, x, y, z, n, step, color;
	//unsigned short int toto;
	float dx, dy, dz, mag, u, xt, yt, zt, d;
	step = wim*wim;
	if (idx < nim) {
		//color = d_im[idx];
		color = 0;
		for (n=0; n<nx1; ++n) {
			
			z = idx / step;
			x = (idx - (z * step));
			y = x / wim;
			x = (x - (y * wim));
			//x1 = d_x1[n];
			//y1 = d_y1[n];
			//z1 = d_z1[n];
			//x1 = tex1Dfetch(tex_x1, n);
			//y1 = tex1Dfetch(tex_y1, n);
			//z1 = tex1Dfetch(tex_z1, n);
			//x2 = d_x2[n];
			//y2 = d_y2[n];
			//z2 = d_z2[n];
			x2 = 20;
			x1 = 10;
			y1 = 10;
			y2 = 10;
			z1 = 20;
			z2 = 10;
			dx = x2-x1;
			dy = y2-y1;
			dz = z2-z1;
			mag = __powf(dx*dx + dy*dy + dz*dz, 0.5);
			u = ((x-x1)*dx + (y-y1)*dy + (z-z1)*dz) / (mag*mag);
			xt = x1 + u*dx;
			yt = y1 + u*dy;
			zt = z1 + u*dz;
			d = __powf((x-xt)*(x-xt) + (y-yt)*(y-yt) + (z-zt)*(z-zt), 0.5);
			//d_im[idx] = d;
			if (d < .707f) {color++;}
			//d = d + 0.5f;
			//d = int(1 / d);
			//color = d;
			//d = 0.5f;
			//color += (x1 + y1 + z1);
			//if (x < 0.707f) {color++;}
			//color += d;
			//__syncthreads();
			}
		d_im[idx] = color;
		
	}
}
*/

__device__ inline void atomicFloatAdd(float* address, float val) {
	int i_val = __float_as_int(val);
	int tmp0 = 0;
	int tmp1;

	while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
	{
		tmp0 = tmp1;
		i_val = __float_as_int(val + __int_as_float(tmp1));
	}

}

// kernel to update image in pet2D EMML algorithm
__global__ void pet2D_im_update(float* im, float* S, float* F, int npix) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < npix && F[idx] != 0.0f) {
		im[idx] = im[idx] / S[idx] * F[idx];
		//im[idx] = F[idx];
	}
}
// kernel to update Q value in pet2D EMML algorithm
__global__ void pet2D_Q_update(int* d_lorval, float* d_Q, int nval) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nval) {
		d_Q[idx] = (float)d_lorval[idx] / d_Q[idx];
	}
}
// kernel to raytrace line in SRM with DDA algorithm
__global__ void pet2D_SRM_DDA(float* d_SRM, int* d_X1, int* d_Y1, int* d_X2, int* d_Y2, int wx, int nx1, int width_image) {
	int length, n, x1, y1, x2, y2, diffx, diffy, LOR_ind;
	float flength, val, x, y, lx, ly, xinc, yinc;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nx1) {
		LOR_ind = idx * wx;
		x1 = d_X1[idx];
		y1 = d_Y1[idx];
		x2 = d_X2[idx];
		y2 = d_Y2[idx];
		diffx = x2-x1;
		diffy = y2-y1;
		lx = abs(diffx);
		ly = abs(diffy);
		length = ly;
		if (lx > length) {length = lx;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		val = 1.0f / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		for (n=0; n<=length; ++n) {
			d_SRM[LOR_ind + (int)y * width_image + (int)x] = val;
			x = x + xinc;
			y = y + yinc;
		}
	}
}
// kernel to raytrace line in SRM with DDA algorithm and ELL sparse matrix format 
__global__ void pet2D_SRM_DDA_ELL(float* d_SRM_vals, int* d_SRM_cols, int* d_x1, int* d_y1, int* d_x2, int* d_y2, int wsrm, int wim, int nx1) {
	int length, n, x1, y1, x2, y2, diffx, diffy, LOR_ind;
	float flength, val, x, y, lx, ly, xinc, yinc;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	val = 1.0f;
	if (idx < nx1) {
		LOR_ind = idx * wsrm;
		x1 = d_x1[idx];
		x2 = d_x2[idx];
		y1 = d_y1[idx];
		y2 = d_y2[idx];
		diffx = x2-x1;
		diffy = y2-y1;
		lx = abs(diffx);
		ly = abs(diffy);
		length = ly;
		if (lx > length) {length = lx;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		//val  = 1.0f / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		for (n=0; n<=length; ++n) {
			d_SRM_vals[LOR_ind + n] = val;
			d_SRM_cols[LOR_ind + n] = (int)y * wim + (int)x;
			x = x + xinc;
			y = y + yinc;
		}
		d_SRM_cols[LOR_ind + n] = -1; // eof
	}
}
/*
// DEV
__global__ void pet3D_IM_SRM_DDA_DEV(unsigned short int* d_x1, unsigned short int* d_y1, unsigned short int* d_z1,
								  unsigned short int* d_x2, unsigned short int* d_y2, unsigned short int* d_z2,
								  int wsrm, int wim, int nx1) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	if (idx < wim) {
		for (i=0;i<nx1;++i) {
			


		}



	}
}
*/


// kernel to raytrace 3D line in SRM with DDA algorithm and ELL sparse matrix format 
__global__ void pet3D_SRM_DDA_ELL(float* d_SRM_vals, int* d_SRM_cols, int wsrm, int wim, int nx1) {
	int length, n, diffx, diffy, diffz, LOR_ind, step;
	float flength, val, x, y, z, lx, ly, lz, xinc, yinc, zinc;
	unsigned short int x1, y1, z1, x2, y2, z2;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	val = 1.0f;
	step = wim*wim;
	
	if (idx < nx1) {
		LOR_ind = idx * wsrm;
		x1 = tex1Dfetch(tex_x1, idx);
		y1 = tex1Dfetch(tex_y1, idx);
		z1 = tex1Dfetch(tex_z1, idx);
		x2 = tex1Dfetch(tex_x2, idx);
		y2 = tex1Dfetch(tex_y2, idx);
		z2 = tex1Dfetch(tex_z2, idx);

		diffx = x2-x1;
		diffy = y2-y1;
		diffz = z2-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		zinc = diffz / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		z = z1 + 0.5f;
		for (n=0; n<=length; ++n) {
			d_SRM_vals[LOR_ind + n] = val;
			d_SRM_cols[LOR_ind + n] = (int)z * step + (int)y * wim + (int)x;
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
		d_SRM_cols[LOR_ind + n] = -1; // eof
	}

}
// kernel to raytrace 3D line in SRM with DDA algorithm on-line
__global__ void pet3D_SRM_DDA_ON(int* d_im, int wim, int nx1, int nim) {

	int length, n, diffx, diffy, diffz, step;
	float flength, x, y, z, lx, ly, lz, xinc, yinc, zinc;
	unsigned short int x1, y1, z1, x2, y2, z2;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	step = wim*wim;
	
	if (idx < nx1) {
		x1 = tex1Dfetch(tex_x1, idx);
		y1 = tex1Dfetch(tex_y1, idx);
		z1 = tex1Dfetch(tex_z1, idx);
		x2 = tex1Dfetch(tex_x2, idx);
		y2 = tex1Dfetch(tex_y2, idx);
		z2 = tex1Dfetch(tex_z2, idx);
		diffx = x2-x1;
		diffy = y2-y1;
		diffz = z2-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = 1.0f / (float)length;
		xinc = diffx * flength;
		yinc = diffy * flength;
		zinc = diffz * flength;
		x = x1;
		y = y1;
		z = z1;
		for (n=0; n<=length; ++n) {
			atomicAdd(&d_im[int(z) * step + (int)y * wim + (int)x], 1);
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
	}
}

// kernel to raytrace 3D line in SRM with DDA algorithm on-line
__global__ void pet3D_SRM_DDA_fixed_ON(int* d_im, int wim, int nx1, int nim) {

	int length, n, diffx, diffy, diffz, step;
	float flength, lx, ly, lz;
	int x, y, z, xinc, yinc, zinc;
	int xt, yt, zt;
	unsigned short int x1, y1, z1, x2, y2, z2;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	step = wim*wim;
	
	if (idx < nx1) {
		x1 = tex1Dfetch(tex_x1, idx);
		y1 = tex1Dfetch(tex_y1, idx);
		z1 = tex1Dfetch(tex_z1, idx);
		x2 = tex1Dfetch(tex_x2, idx);
		y2 = tex1Dfetch(tex_y2, idx);
		z2 = tex1Dfetch(tex_z2, idx);
		diffx = x2-x1;
		diffy = y2-y1;
		diffz = z2-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = 1.0f / (float)length;
		xinc = (int)(diffx * flength * 8388608);
		yinc = (int)(diffy * flength * 8388608);
		zinc = (int)(diffz * flength * 8388608);
		x = (int)(x1 * 8388608);
		y = (int)(y1 * 8388608);
		z = (int)(z1 * 8388608);
		for (n=0; n<=length; ++n) {
			xt = x;
			yt = y;
			zt = z;
			atomicAdd(&d_im[(zt >> 23) * step + (yt >> 23) * wim + (xt >> 23)], 1);
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
	}
}


// kernel to raytrace 3D line in SRM with DDA algorithm and compute F on-line
__global__ void pet3D_SRM_DDA_F_ON(unsigned int* d_F, int wim, int nx1, int nim, float scale) {

	int length, n, diffx, diffy, diffz, step;
	float flength, x, y, z, lx, ly, lz, xinc, yinc, zinc, Qi;
	unsigned short int x1, y1, z1;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	step = wim*wim;
	
	if (idx < nx1) {
		Qi = 0.0f;
		x1 = tex1Dfetch(tex_x1, idx);
		y1 = tex1Dfetch(tex_y1, idx);
		z1 = tex1Dfetch(tex_z1, idx);
		diffx = tex1Dfetch(tex_x2, idx)-x1;
		diffy = tex1Dfetch(tex_y2, idx)-y1;
		diffz = tex1Dfetch(tex_z2, idx)-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = 1.0f / (float)length;
		xinc = diffx * flength;
		yinc = diffy * flength;
		zinc = diffz * flength;
		x = x1;
		y = y1;
		z = z1;
		for (n=0; n<=length; ++n) {
			Qi = Qi + tex1Dfetch(tex_im, (int)z * step + (int)y * wim + (int)x);
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}

		// compute F
		if (Qi==0.0f) {return;}
		Qi = 1 / Qi;
		x = x1;
		y = y1;
		z = z1;
		for (n=0; n<=length; ++n) {
			//atomicFloatAdd(&d_F[(int)z * step + (int)y * wim + (int)x], Qi);
			atomicAdd(&d_F[(int)z * step + (int)y * wim + (int)x], (unsigned int)(Qi*scale));
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
	}
}

// Same as pet3D_SRM_DDA_F_ON with attenuation correction
__global__ void pet3D_SRM_DDA_F_ATT_ON(unsigned int* d_F, int wim, int nx1, int nim, float scale) {

	int length, n, diffx, diffy, diffz, step, ind;
	float flength, x, y, z, lx, ly, lz, xinc, yinc, zinc, Qi, Ai;
	unsigned short int x1, y1, z1;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	step = wim*wim;
	
	if (idx < nx1) {
		Qi = 0.0f;
		Ai = 0.0f;
		x1 = tex1Dfetch(tex_x1, idx);
		y1 = tex1Dfetch(tex_y1, idx);
		z1 = tex1Dfetch(tex_z1, idx);
		diffx = tex1Dfetch(tex_x2, idx)-x1;
		diffy = tex1Dfetch(tex_y2, idx)-y1;
		diffz = tex1Dfetch(tex_z2, idx)-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = 1.0f / (float)length;
		xinc = diffx * flength;
		yinc = diffy * flength;
		zinc = diffz * flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		z = z1 + 0.5f;
		for (n=0; n<=length; ++n) {
			ind = (int)z * step + (int)y * wim + (int)x;
			Qi = Qi + tex1Dfetch(tex_im, ind);
			Ai = Ai - tex1Dfetch(tex_mumap, ind);
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
		// compute F
		if (Qi==0.0f) {return;}
		if (Ai < -5.0f) {Ai = -5.0f;}
		Qi = Qi * __expf(Ai);
		Qi = 1 / Qi;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		z = z1 + 0.5f;
		for (n=0; n<=length; ++n) {
			//atomicFloatAdd(&d_F[(int)z * step + (int)y * wim + (int)x], Qi);
			atomicAdd(&d_F[(int)z * step + (int)y * wim + (int)x], (unsigned int)(Qi*scale));
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
	}

}
// kernel to raytrace 3D line in SRM with DDA algorithm and ELL sparse matrix format 
__global__ void pet3D_SRM_DDA_ELL_Q(float* d_SRM_vals, int* d_SRM_cols, float* d_im, float* d_Q,
									unsigned short int* d_x1, unsigned short int* d_y1, unsigned short int* d_z1,
									unsigned short int* d_x2, unsigned short int* d_y2, unsigned short int* d_z2,
									int wsrm, int wim, int nx1) {
	int length, n, x1, y1, z1, x2, y2, z2, diffx, diffy, diffz, LOR_ind, step, vcol;
	float flength, val, x, y, z, lx, ly, lz, xinc, yinc, zinc, Qi;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	val = 1.0f;
	step = wim*wim;
	Qi = 0.0f;
	
	if (idx < nx1) {
		LOR_ind = idx * wsrm;
		x1 = d_x1[idx];
		x2 = d_x2[idx];
		y1 = d_y1[idx];
		y2 = d_y2[idx];
		z1 = d_z1[idx];
		z2 = d_z2[idx];
		diffx = x2-x1;
		diffy = y2-y1;
		diffz = z2-z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		zinc = diffz / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		z = z1 + 0.5f;
		for (n=0; n<=length; ++n) {
			d_SRM_vals[LOR_ind + n] = val;
			vcol = (int)z * step + (int)y * wim + (int)x;
			d_SRM_cols[LOR_ind + n] = vcol;
			Qi = Qi + d_im[vcol];
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
		d_SRM_cols[LOR_ind + n] = -1; // eof
		d_Q[idx] = Qi;
	}

}
// kernel to raytrace line in SRM with DDA anti-aliased version 2 pix, SRM is in ELL sparse matrix format 
__global__ void pet2D_SRM_DDAA_ELL(float* d_SRM_vals, int* d_SRM_cols, int* d_x1, int* d_y1, int* d_x2, int* d_y2, int wsrm, int wim, int nx1) {
	int length, n, x1, y1, x2, y2, diffx, diffy, LOR_ind, ind, ind2, xint, yint;
	float flength, val, vu, x, y, lx, ly, xinc, yinc;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nx1) {
		LOR_ind = idx * wsrm;
		x1 = d_x1[idx];
		x2 = d_x2[idx];
		y1 = d_y1[idx];
		y2 = d_y2[idx];
		diffx = x2-x1;
		diffy = y2-y1;
		lx = abs(diffx);
		ly = abs(diffy);
		length = ly;
		if (lx > length) {length = lx;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		// first pixel
		xint = int(x);
		yint = int(y);
		val = 1 - fabs(x - (xint + 0.5f));
		d_SRM_vals[LOR_ind] = val;
		d_SRM_cols[LOR_ind] = yint * wim + xint;
		x = x + xinc;
		y = y + yinc;
		// line
		for (n=1; n<length; ++n) {
			xint = int(x);
			yint = int(y);
			ind = yint * wim + xint;
			val = 1 - fabs(x - (xint + 0.5f));
			vu = (x - xint) * 0.5f;
			// vd = 0.5 - vu;
			ind2 = LOR_ind + 2*n;
			d_SRM_vals[ind2] = vu;
			d_SRM_cols[ind2] = ind + 1;
			d_SRM_vals[ind2 + 1] = val;
			d_SRM_cols[ind2 + 1] = ind;
			x = x + xinc;
			y = y + yinc;
		}
		// last pixel
		xint = int(x);
		yint = int(y);
		val = 1 - fabs(x - (xint + 0.5f));
		ind2 = LOR_ind + 2*n;
		d_SRM_vals[ind2] = val;
		d_SRM_cols[ind2] = yint * wim + xint;
	}
}
// init SRM to zeros with the format ELL
__global__ void pet2D_SRM_ELL_init(float* d_SRM_vals, int* d_SRM_cols, int wsrm, int nx) {
	int j, ind;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nx) {
		// due to limitation of gridsize < 65536
		for (j=0; j<wsrm; ++j) {
			ind = idx * wsrm + j;
			d_SRM_vals[ind] = 0.0f;
			d_SRM_cols[ind] = -1;
		}
	}
}
// init Q and F to zeros
__global__ void pet2D_QF_init(float* d_Q, float* d_F, int nq, int nf) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nq) {d_Q[idx] = 0.0f;}
	if (idx < nf) {d_F[idx] = 0.0f;}
}
// init Q to zeros
__global__ void pet2D_Q_init(float* d_Q, int nq) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nq) {d_Q[idx] = 0.0f;}
}
// Compute Q vector by SRM * IM (ELL sparse matrix format)
__global__ void pet2D_ell_spmv(float* d_SRM_vals, int* d_SRM_cols, float* d_Q, float* d_im,  int niv, int njv) {
	int j, ind, vcol;
	float sum;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < niv) {
		ind = idx * njv;
		vcol = d_SRM_cols[ind];
		j = 0;
		sum = 0.0f;
		while (vcol != -1) {
			sum += (d_SRM_vals[ind+j] * d_im[vcol]);
			++j;
			vcol = d_SRM_cols[ind+j];
		}
		d_Q[idx] = sum;
	}
}
// Compute F vector by SRM^T / Q (ELL sparse matrix format)
__global__ void pet2D_ell_F(float* d_SRM_vals, int* d_SRM_cols, float* d_F, float* d_Q, int niv, int njv) {
	int i, ind, vcol;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float Qi;
	if (idx < njv) {
		for (i=0; i < niv; ++i) {
			Qi = d_Q[i];
			if (Qi==0.0f) {continue;}
			ind = i * njv + idx;
			vcol = d_SRM_cols[ind];
			if (vcol != -1) {d_F[vcol] += (d_SRM_vals[ind] / Qi);}
			__syncthreads();
		}
	}
}

// Compute col sum of ELL matrix (to get im from SRM)
__global__ void matrix_ell_sumcol(float* d_vals, int niv, int njv, int* d_cols, float* d_im) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i, vcol, ind;
	if (idx < njv) {
		for (i=0; i<niv; ++i) {
			ind = i * njv + idx;
			vcol = d_cols[ind];
			if (vcol != -1) {d_im[vcol] += d_vals[ind];}
			__syncthreads();
		}
	}
}


__global__ void matrix_ell_spmv(float* d_vals, int* d_cols, float* d_res, int niv, int njv) {
	int j, ind, vcol;
	float sum;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < niv) {
		ind = idx * njv;
		vcol = d_cols[ind];
		j = 0;
		sum = 0.0f;
		while (vcol != -1) {
			sum += (d_vals[ind+j] * tex1D(tex1, vcol));
			++j;
			vcol = d_cols[ind+j];
		}
		d_res[idx] = sum;
	}
}

// Perform a multiplication between a complex and a real vectors
__global__ void vector_complex_x_real(hipfftComplex* dcpx, float* dr, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float r, j, h;
	if (idx < n) {
		r = dcpx[idx].x;
		j = dcpx[idx].y;
		h = dr[idx];
		r = r * h;
		j = j * h;
		dcpx[idx].x = r;
		dcpx[idx].y = j;
	}
}

// Perform a mulitplication between a real vectors and an alpha value
__global__ void vector_real_x_cst(float* dr, float alpha, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float val;
	if (idx < n) {
		val = dr[idx];
		val = val * alpha;
		dr[idx] = val;
	}
}

void kernel_pet2D_EMML_wrap_cuda(float* SRM, int nlor, int npix, float* im, int npixim, int* LOR_val, int nval, float* S, int ns, int maxit) {
	// select a GPU
	hipSetDevice(0);
	// init cublas
	hipblasStatus_t status;
	status = cublasInit();
	// allocate device memory for SRM, im, Q and F
	int size_SRM = nlor * npix;
	float* d_SRM;
	float* d_im;
	float* d_Q;
	float* d_F;
	float* d_S;
	int* d_lorval;
	status = cublasAlloc(size_SRM, sizeof(float), (void**)&d_SRM);
	status = cublasAlloc(npixim, sizeof(float), (void**)&d_im);
	status = cublasAlloc(nlor, sizeof(float), (void**)&d_Q);
	status = cublasAlloc(nlor, sizeof(float), (void**)&d_F);
	status = cublasAlloc(ns, sizeof(float), (void**)&d_S);
	status = cublasAlloc(nval, sizeof(int), (void**)&d_lorval);
	// load SRM, SM, LOR_val and im to the device
	status = hipblasSetVector(size_SRM, sizeof(float), SRM, 1, d_SRM, 1);
	status = hipblasSetVector(npixim, sizeof(float), im, 1, d_im, 1);
	status = hipblasSetVector(ns, sizeof(float), S, 1, d_S, 1);
	status = hipblasSetVector(nval, sizeof(int), LOR_val, 1, d_lorval, 1);
	if (status != 0) {exit(0);}
	int ite, block_size1, grid_size1, block_size2, grid_size2;
	block_size1 = 256;
	grid_size1 = (nlor + block_size1 - 1) / block_size1;
	block_size2 = 64;
	grid_size2 = (npix + block_size2 - 1) / block_size2;
	dim3 threads1(block_size1);
	dim3 grid1(grid_size1);
	dim3 threads2(block_size2);
	dim3 grid2(grid_size2);
	for (ite=0; ite<maxit; ++ite) {
		// compute Q
		hipblasSgemv('t', npix, nlor, 1.0, d_SRM, npix, d_im, 1, 0.0, d_Q, 1);
		pet2D_Q_update<<< grid1, threads1 >>>(d_lorval, d_Q, nval);
		// compute f = sum{SRMi / qi} for each i LOR
		hipblasSgemv('n', npix, nlor, 1.0, d_SRM, npix, d_Q, 1, 0.0, d_F, 1);
		// update image
		pet2D_im_update<<< grid2, threads2 >>>(d_im, d_S, d_F, npix);
	}
	// get results
	status = cublasGetError();
	status = hipblasGetVector(npix, sizeof(float), d_im, 1, im, 1);
	// free memory
	status = cublasFree(d_SRM);
	status = cublasFree(d_im);
	status = cublasFree(d_Q);
	status = cublasFree(d_F);
	status = cublasFree(d_S);
	// prepare to quit
	status = cublasShutdown();
}

void kernel_pet2D_SRM_DDA_wrap_cuda(float* SRM, int wy, int wx, int* X1, int nx1, int* Y1, int ny1, int* X2, int nx2, int* Y2, int ny2, int width_image) {
	// select a GPU
	hipSetDevice(0);
	// some vars
	int size_SRM = wy * wx;
	unsigned int mem_size_SRM = sizeof(float) * size_SRM;
	unsigned int mem_size_point = sizeof(int) * nx1;
	// alloacte device memory for SRM, x1, y1, x2, and y2
	float* d_SRM;
	int* d_X1;
	int* d_Y1;
	int* d_X2;
	int* d_Y2;
	hipMalloc((void**) &d_SRM, mem_size_SRM);
	hipMemset(d_SRM, 0.0f, mem_size_SRM);
	hipMalloc((void**) &d_X1, mem_size_point);
	hipMalloc((void**) &d_Y1, mem_size_point);
	hipMalloc((void**) &d_X2, mem_size_point);
	hipMalloc((void**) &d_Y2, mem_size_point);
	// copy host memory to device
	hipMemcpy(d_X1, X1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_Y1, Y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_X2, X2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_Y2, Y2, mem_size_point, hipMemcpyHostToDevice);
	// setup execution parameters
	int block_size, grid_size;
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	dim3 threads(block_size);
	dim3 grid(grid_size);

	//timeval start, end;
	//double t1, t2, diff;
	//gettimeofday(&start, NULL);
	//t1 = start.tv_sec + start.tv_usec / 1000000.0;
	// DDA kernel
	pet2D_SRM_DDA<<< grid, threads >>>(d_SRM, d_X1, d_Y1, d_X2, d_Y2, wx, nx1, width_image);
	hipDeviceSynchronize();
	// get back results to the host
	hipMemcpy(SRM, d_SRM, mem_size_SRM, hipMemcpyDeviceToHost);
	//gettimeofday(&end, NULL);
	//t2 = end.tv_sec + end.tv_usec / 1000000.0;
	//diff = t2 - t1;
	//printf("time %f s\n", diff);
	// clean up memory
	hipFree(d_SRM);
	hipFree(d_X1);
	hipFree(d_Y1);
	hipFree(d_X2);
	hipFree(d_Y2);
}

void kernel_matrix_ell_spmv_wrap_cuda(float* vals, int niv, int njv, int* cols, int nic, int njc, float* y, int ny, float* res, int nres) {
	// select a GPU
	hipSetDevice(0);
	// some vars
	int size_data = niv * njv;
	unsigned int mem_size_dataf = sizeof(float) * size_data;
	unsigned int mem_size_y = sizeof(float) * ny;
	unsigned int mem_size_res = sizeof(float) * nres;
	unsigned int mem_size_datai = sizeof(int) * size_data;
	// alloacte device memory
	float* d_vals;
	float* d_res;
	//float* d_y;
	int* d_cols;
	hipMalloc((void**) &d_vals, mem_size_dataf);
	hipMalloc((void**) &d_res, mem_size_res);
	//hipMalloc((void**) &d_y, mem_size_y);
	hipMalloc((void**) &d_cols, mem_size_datai);
	// copy host memory to device
	hipMemcpy(d_vals, vals, mem_size_dataf, hipMemcpyHostToDevice);
	//hipMemcpy(d_y, y, mem_size_res, hipMemcpyHostToDevice);
	hipMemcpy(d_cols, cols, mem_size_datai, hipMemcpyHostToDevice);
	// prepare texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    hipMallocArray( &cu_array, &channelDesc, ny, 1 ); 
    hipMemcpyToArray(cu_array, 0, 0, y, mem_size_y, hipMemcpyHostToDevice);
	tex1.addressMode[0] = hipAddressModeClamp;
    tex1.addressMode[1] = hipAddressModeClamp;
    tex1.filterMode = hipFilterModePoint;
    tex1.normalized = false;
    hipBindTextureToArray(tex1, cu_array, channelDesc);

	// setup execution parameters
	int block_size, grid_size;
	block_size = 256;
	grid_size = (niv + block_size - 1) / block_size;
	dim3 threads(block_size);
	dim3 grid(grid_size);
	timeval start, end;
	double t1, t2, diff;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	// spmv kernel
	matrix_ell_spmv<<< grid, threads >>>(d_vals, d_cols, d_res, niv, njv);
	hipDeviceSynchronize();
	// get back results to the host
	hipMemcpy(res, d_res, mem_size_res, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel time %f s\n", diff);
	// clean up memory
	hipFree(d_vals);
	hipFree(d_cols);
	//hipFree(d_y);
	hipFree(d_res);
}

void kernel_pet2D_LM_EMML_DDA_ELL_wrap_cuda(int* x1, int nx1, int* y1, int ny1, int* x2, int nx2, int* y2, int ny2, float* im, int nim, float* S, int ns, int wsrm, int wim, int maxite) {
	// select a GPU
	hipSetDevice(0);
	// to time
	timeval start, end;
	double t1, t2, diff;
	// vars
	int ite;
	int block_size, grid_size;
	dim3 threads, grid;
	dim3 threads2, grid2;
	dim3 threads3, grid3;
	// allocate device memory
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	int size_SRM = nx1 * wsrm;
	unsigned int mem_size_iSRM = size_SRM * sizeof(int);
	unsigned int mem_size_fSRM = size_SRM * sizeof(float);
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_S = ns * sizeof(float);
	unsigned int mem_size_Q = nx1 * sizeof(float);
	unsigned int mem_size_F = nim * sizeof(float);
	unsigned int mem_size_point = nx1 * sizeof(int);
	printf("mem tot %i\n", 4*(size_SRM + size_SRM + nim + ns + nx1 + nim + 4*nx1));
	float* d_SRM_vals;
	int* d_SRM_cols;
	float* d_im;
	float* d_S;
	float* d_Q;
	float* d_F;
	int* d_x1;
	int* d_x2;
	int* d_y1;
	int* d_y2;
	hipMalloc((void**) &d_SRM_vals, mem_size_fSRM);
	hipMalloc((void**) &d_SRM_cols, mem_size_iSRM);
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_S, mem_size_S);
	hipMalloc((void**) &d_Q, mem_size_Q);
	hipMalloc((void**) &d_F, mem_size_F);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_S, S, mem_size_S, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("prepare mem: %f s\n", diff);

	// Init kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	pet2D_SRM_ELL_init<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, nx1);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel SRM init: %f s\n", diff);
	// DDA kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	pet2D_SRM_DDA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_x1, d_y1, d_x2, d_y2, wsrm, wim, nx1);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel DDA: %f s\n", diff);
	// IM kernel
	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	matrix_ell_sumcol<<<grid, threads>>>(d_SRM_vals, nx1, wsrm, d_SRM_cols, d_im);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel compute IM: %f s\n", diff);

	// Iteration loop
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;

	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads2.x = block_size;
	grid2.x = grid_size;
	
	block_size = 64;
	grid_size = (nim + block_size - 1) / block_size;
	threads3.x = block_size;
	grid3.x = grid_size;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	for (ite=0; ite<maxite; ++ite) {
		// init F and Q to zeros
		pet2D_QF_init<<<grid, threads>>>(d_Q, d_F, nx1, nim);
		// compute Q
		pet2D_ell_spmv<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_Q, d_im, nx1, wsrm);
		// compute f = sum{SRMi / qi} for each i LOR
		pet2D_ell_F<<<grid2, threads2>>>(d_SRM_vals, d_SRM_cols, d_F, d_Q, nx1, wsrm);
		// update image
		pet2D_im_update<<< grid3, threads3 >>>(d_im, d_S, d_F, nim);

	}

	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel iter: %f s\n", diff);
	

	// get back image
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("get image back: %f s\n", diff);

	hipFree(d_SRM_vals);
	hipFree(d_SRM_cols);
	hipFree(d_im);
	hipFree(d_S);
	hipFree(d_Q);
	hipFree(d_F);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_x2);
	hipFree(d_y2);
	
}

// Compute the first image in LM 2D-OSEM algorithm (from x, y build SRM, then compute IM)
void kernel_pet2D_IM_SRM_DDA_ELL_wrap_cuda(int* x1, int nx1, int* y1, int ny1, int* x2, int nx2, int* y2, int ny2, float* im, int nim, int wsrm, int wim) {
	// select a GPU
	hipSetDevice(0);
	// vars
	int block_size, grid_size;
	dim3 threads, grid;
	// allocate device memory
	int size_SRM = nx1 * wsrm;
	unsigned int mem_size_iSRM = size_SRM * sizeof(int);
	unsigned int mem_size_fSRM = size_SRM * sizeof(float);
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_point = nx1 * sizeof(int);
	float* d_SRM_vals;
	int* d_SRM_cols;
	float* d_im;
	int* d_x1;
	int* d_x2;
	int* d_y1;
	int* d_y2;
	hipMalloc((void**) &d_SRM_vals, mem_size_fSRM);
	hipMalloc((void**) &d_SRM_cols, mem_size_iSRM);
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	// Init kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_ELL_init<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, nx1);
	// DDA kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_DDA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_x1, d_y1, d_x2, d_y2, wsrm, wim, nx1);
	//pet2D_SRM_DDAA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_x1, d_y1, d_x2, d_y2, wsrm, wim, nx1);
	// IM kernel
	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	matrix_ell_sumcol<<<grid, threads>>>(d_SRM_vals, nx1, wsrm, d_SRM_cols, d_im);
	// get back image
	hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
	// Free mem
	hipFree(d_SRM_vals);
	hipFree(d_SRM_cols);
	hipFree(d_im);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_x2);
	hipFree(d_y2);
}

// Update image for the 2D-LM-OSEM reconstruction (from x, y, IM and S, build SRM in ELL format then update IM)
void kernel_pet2D_IM_SRM_DDA_ELL_iter_wrap_cuda(int* x1, int nx1, int* y1, int ny1, int* x2, int nx2, int* y2, int ny2, float* S, int ns, float* im, int nim, int wsrm, int wim) {
	// select a GPU
	hipSetDevice(0);
	// vars
	int block_size, grid_size;
	dim3 threads, grid;
	dim3 threads2, grid2;
	dim3 threads3, grid3;
	// allocate device memory
	int size_SRM = nx1 * wsrm;
	unsigned int mem_size_iSRM = size_SRM * sizeof(int);
	unsigned int mem_size_fSRM = size_SRM * sizeof(float);
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_S = ns * sizeof(float);
	unsigned int mem_size_Q = nx1 * sizeof(float);
	unsigned int mem_size_F = nim * sizeof(float);
	unsigned int mem_size_point = nx1 * sizeof(int);
	float* d_SRM_vals;
	int* d_SRM_cols;
	float* d_im;
	float* d_S;
	float* d_Q;
	float* d_F;
	int* d_x1;
	int* d_x2;
	int* d_y1;
	int* d_y2;
	hipMalloc((void**) &d_SRM_vals, mem_size_fSRM);
	hipMalloc((void**) &d_SRM_cols, mem_size_iSRM);
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_S, mem_size_S);
	hipMalloc((void**) &d_Q, mem_size_Q);
	hipMalloc((void**) &d_F, mem_size_F);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_S, S, mem_size_S, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	// Init kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_ELL_init<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, nx1);
	// DDA kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_DDA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_x1, d_y1, d_x2, d_y2, wsrm, wim, nx1);
	//pet2D_SRM_DDAA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_x1, d_y1, d_x2, d_y2, wsrm, wim, nx1);
	// One iteration
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads2.x = block_size;
	grid2.x = grid_size;
	block_size = 64;
	grid_size = (nim + block_size - 1) / block_size;
	threads3.x = block_size;
	grid3.x = grid_size;
	// init F and Q to zeros
	pet2D_QF_init<<<grid, threads>>>(d_Q, d_F, nx1, nim);
	// compute Q
	pet2D_ell_spmv<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_Q, d_im, nx1, wsrm);
	// compute f = sum{SRMi / qi} for each i LOR
	pet2D_ell_F<<<grid2, threads2>>>(d_SRM_vals, d_SRM_cols, d_F, d_Q, nx1, wsrm);
	// update image
	pet2D_im_update<<< grid3, threads3 >>>(d_im, d_S, d_F, nim);
	// get back image
	hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
	// Free mem
	hipFree(d_SRM_vals);
	hipFree(d_SRM_cols);
	hipFree(d_im);
	hipFree(d_S);
	hipFree(d_Q);
	hipFree(d_F);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_x2);
	hipFree(d_y2);
}

// Compute the first image in LM 3D-OSEM algorithm (from x, y build SRM, then compute IM)
void kernel_pet3D_IM_SRM_DDA_ELL_wrap_cuda(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1, unsigned short int* z1, int nz1,
										   unsigned short int* x2, int nx2, unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
										   float* im, int nim, int wsrm, int wim, int ID) {
	// select a GPU
	if (ID != -1) {hipSetDevice(ID);}
	// vars
	int block_size, grid_size;
	dim3 threads, grid;
	// allocate device memory
	int size_SRM = nx1 * wsrm;
	unsigned int mem_size_iSRM = size_SRM * sizeof(int);
	unsigned int mem_size_fSRM = size_SRM * sizeof(float);
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_point = nx1 * sizeof(unsigned short int);
	float* d_SRM_vals;
	int* d_SRM_cols;
	float* d_im;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	hipMalloc((void**) &d_SRM_vals, mem_size_fSRM);
	hipMalloc((void**) &d_SRM_cols, mem_size_iSRM);
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_z1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	hipMalloc((void**) &d_z2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z1, z1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z2, z2, mem_size_point, hipMemcpyHostToDevice);
	// Init textures
	hipBindTexture(NULL, tex_x1, d_x1, mem_size_point);
	hipBindTexture(NULL, tex_y1, d_y1, mem_size_point);
	hipBindTexture(NULL, tex_z1, d_z1, mem_size_point);
	hipBindTexture(NULL, tex_x2, d_x2, mem_size_point);
	hipBindTexture(NULL, tex_y2, d_y2, mem_size_point);
	hipBindTexture(NULL, tex_z2, d_z2, mem_size_point);
	// Init kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_ELL_init<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, nx1);
	// DDA kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet3D_SRM_DDA_ELL<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, wim, nx1);
	// IM kernel
	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	matrix_ell_sumcol<<<grid, threads>>>(d_SRM_vals, nx1, wsrm, d_SRM_cols, d_im);
	// get back image
	hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
	// Free mem
	hipFree(d_SRM_vals);
	hipFree(d_SRM_cols);
	hipFree(d_im);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_z1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_z2);
}

// Update image for the 3D-LM-OSEM reconstruction (from x, y, IM and S, build SRM in ELL format then return F)
void kernel_pet3D_IM_SRM_DDA_ELL_iter_wrap_cuda(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1, unsigned short int* z1, int nz1,
												unsigned short int* x2, int nx2, unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
												float* im, int nim, float* F, int nf, int wsrm, int wim, int ID){

	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// vars
	int block_size, grid_size;
	dim3 threads, grid;
	dim3 threads2, grid2;
	dim3 threads3, grid3;
	// allocate device memory
	int size_SRM = nx1 * wsrm;
	unsigned int mem_size_iSRM = size_SRM * sizeof(int);
	unsigned int mem_size_fSRM = size_SRM * sizeof(float);
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_Q = nx1 * sizeof(float);
	unsigned int mem_size_F = nim * sizeof(float);
	unsigned int mem_size_point = nx1 * sizeof(unsigned short int);
	float* d_SRM_vals;
	int* d_SRM_cols;
	float* d_im;
	float* d_Q;
	float* d_F;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	hipMalloc((void**) &d_SRM_vals, mem_size_fSRM);
	hipMalloc((void**) &d_SRM_cols, mem_size_iSRM);
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_Q, mem_size_Q);
	hipMalloc((void**) &d_F, mem_size_F);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_z1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	hipMalloc((void**) &d_z2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_F, F, mem_size_F, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z1, z1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z2, z2, mem_size_point, hipMemcpyHostToDevice);
	// Init kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_SRM_ELL_init<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, wsrm, nx1);
	// DDA kernel
	pet3D_SRM_DDA_ELL_Q<<<grid, threads>>>(d_SRM_vals, d_SRM_cols, d_im, d_Q, d_x1, d_y1, d_z1, d_x2, d_y2, d_z2, wsrm, wim, nx1);
	/*
	// init Q to zeros
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet2D_Q_init<<<grid, threads>>>(d_Q, nx1);
	// compute Q
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads2.x = block_size;
	grid2.x = grid_size;
	pet2D_ell_spmv<<<grid2, threads2>>>(d_SRM_vals, d_SRM_cols, d_Q, d_im, nx1, wsrm);
	*/
	// compute f = sum{SRMi / qi} for each i LOR
	block_size = 8;
	grid_size = (wsrm + block_size - 1) / block_size;
	threads3.x = block_size;
	grid3.x = grid_size;
	pet2D_ell_F<<<grid3, threads3>>>(d_SRM_vals, d_SRM_cols, d_F, d_Q, nx1, wsrm);
	// get back F
	hipMemcpy(F, d_F, mem_size_F, hipMemcpyDeviceToHost);

	// Free mem
	hipFree(d_SRM_vals);
	hipFree(d_SRM_cols);
	hipFree(d_im);
	hipFree(d_Q);
	hipFree(d_F);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_z1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_z2);
}

/***********************************************
 * USED
 ***********************************************/

// Compute the first image in LM 3D-OSEM algorithm (from x, y build SRM, then compute IM)
void kernel_pet3D_IM_SRM_DDA_wrap_cuda(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1,
									   unsigned short int* z1, int nz1, unsigned short int* x2, int nx2,
									   unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
									   int* im, int nim1, int nim2, int nim3, int wim, int ID) {
	// select a GPU
	if (ID != -1) {hipSetDevice(ID);}
	// vars
	int block_size, grid_size;
	dim3 threads, grid;
	// allocate device memory
	int nim = nim1 * nim2 * nim3;
	unsigned int mem_size_im = nim * sizeof(int);
	unsigned int mem_size_point = nx1 * sizeof(unsigned short int);
	int* d_im;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_z1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	hipMalloc((void**) &d_z2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z1, z1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z2, z2, mem_size_point, hipMemcpyHostToDevice);
	// texture
	hipBindTexture(NULL, tex_x1, d_x1, mem_size_point);
	hipBindTexture(NULL, tex_y1, d_y1, mem_size_point);
	hipBindTexture(NULL, tex_z1, d_z1, mem_size_point);
	hipBindTexture(NULL, tex_x2, d_x2, mem_size_point);
	hipBindTexture(NULL, tex_y2, d_y2, mem_size_point);
	hipBindTexture(NULL, tex_z2, d_z2, mem_size_point);
	// IM kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16 Mlines
	threads.x = block_size;
	grid.x = grid_size;
	pet3D_SRM_DDA_ON<<<grid, threads>>>(d_im, wim, nx1, nim);
	//pet3D_SRM_DDA_fixed_ON<<<grid, threads>>>(d_im, wim, nx1, nim);
	// get back image
	hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
	// Free mem
	hipFree(d_im);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_z1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_z2);
	hipDeviceReset();
}

// Compute update in LM 3D-OSEM algorithm on-line with DDA line drawing
void kernel_pet3D_IM_SRM_DDA_ON_iter_wrap_cuda(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1,
											   unsigned short int* z1, int nz1,	unsigned short int* x2, int nx2,
											   unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
											   float* im, int nim1, int nim2, int nim3, float* F, int nf1, int nf2, int nf3,
											   int wim, int ID){

	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// vars
	int block_size, grid_size, i;
	dim3 threads, grid;
	int nim = nim1 * nim2 * nim3;
	// Need to change
	int* Fi = (int*)calloc(nim, sizeof(int));
	// allocate device memory
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_F = nim * sizeof(unsigned int);
	unsigned int mem_size_point = nx1 * sizeof(unsigned short int);
	float* d_im;
	unsigned int* d_F;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_F, mem_size_F);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_z1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	hipMalloc((void**) &d_z2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_F, Fi, mem_size_F, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z1, z1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z2, z2, mem_size_point, hipMemcpyHostToDevice);
	// prepare texture
	hipBindTexture(NULL, tex_im, d_im, mem_size_im);
	hipBindTexture(NULL, tex_x1, d_x1, mem_size_point);
	hipBindTexture(NULL, tex_y1, d_y1, mem_size_point);
	hipBindTexture(NULL, tex_z1, d_z1, mem_size_point);
	hipBindTexture(NULL, tex_x2, d_x2, mem_size_point);
	hipBindTexture(NULL, tex_y2, d_y2, mem_size_point);
	hipBindTexture(NULL, tex_z2, d_z2, mem_size_point);
	// float to int scale
	float scale = 4000.0f;
	// kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16e6 lines
	threads.x = block_size;
	grid.x = grid_size;
	pet3D_SRM_DDA_F_ON<<<grid, threads>>>(d_F, wim, nx1, nim, scale);
	// get back F and convert
	hipMemcpy(Fi, d_F, nim*sizeof(float), hipMemcpyDeviceToHost);
	scale = 1 / scale;
	for (i=0; i<nim; ++i) {F[i] = (float)Fi[i] * scale;}
	// Free mem
	free(Fi);
	hipFree(d_im);
	hipFree(d_F);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_z1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_z2);
	hipDeviceReset();
}

// DEV Compute update in LM 3D-OSEM algorithm on-line with DDA line drawing and attenuation
void kernel_pet3D_IM_ATT_SRM_DDA_ON_iter_wrap_cuda(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1,
												   unsigned short int* z1, int nz1,	unsigned short int* x2, int nx2,
												   unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
												   float* im, int nim1, int nim2, int nim3,
												   float* F, int nf1, int nf2, int nf3,
												   float* mumap, int nmu1, int nmu2, int nmu3, int wim, int ID){

	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// vars
	int block_size, grid_size, i;
	dim3 threads, grid;
	int nim = nim1 * nim2 * nim3;
	// Need to change
	int* Fi = (int*)calloc(nim, sizeof(int));
	// allocate device memory
	unsigned int mem_size_im = nim * sizeof(float);
	unsigned int mem_size_F = nim * sizeof(unsigned int);
	unsigned int mem_size_point = nx1 * sizeof(unsigned short int);
	float* d_im;
	float* d_mumap;
	unsigned int* d_F;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	hipMalloc((void**) &d_im, mem_size_im);
	hipMalloc((void**) &d_mumap, mem_size_im);
	hipMalloc((void**) &d_F, mem_size_F);
	hipMalloc((void**) &d_x1, mem_size_point);
	hipMalloc((void**) &d_y1, mem_size_point);
	hipMalloc((void**) &d_z1, mem_size_point);
	hipMalloc((void**) &d_x2, mem_size_point);
	hipMalloc((void**) &d_y2, mem_size_point);
	hipMalloc((void**) &d_z2, mem_size_point);
	// copy from host to device
	hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_mumap, mumap, mem_size_im, hipMemcpyHostToDevice);
	hipMemcpy(d_F, Fi, mem_size_F, hipMemcpyHostToDevice);
	hipMemcpy(d_x1, x1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z1, z1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_x2, x2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_y2, y2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_z2, z2, mem_size_point, hipMemcpyHostToDevice);
	// prepare texture
	hipBindTexture(NULL, tex_im, d_im, mem_size_im);
	hipBindTexture(NULL, tex_mumap, d_mumap, mem_size_im);
	hipBindTexture(NULL, tex_x1, d_x1, mem_size_point);
	hipBindTexture(NULL, tex_y1, d_y1, mem_size_point);
	hipBindTexture(NULL, tex_z1, d_z1, mem_size_point);
	hipBindTexture(NULL, tex_x2, d_x2, mem_size_point);
	hipBindTexture(NULL, tex_y2, d_y2, mem_size_point);
	hipBindTexture(NULL, tex_z2, d_z2, mem_size_point);
	// float to int scale
	float scale = 4000.0f;
	// kernel
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size; // CODE IS LIMITED TO < 16e6 lines
	threads.x = block_size;
	grid.x = grid_size;
	//pet3D_SRM_DDA_F_ON<<<grid, threads>>>(d_F, wim, nx1, nim, scale);
	pet3D_SRM_DDA_F_ATT_ON<<<grid, threads>>>(d_F, wim, nx1, nim, scale);
	// get back F and convert
	hipMemcpy(Fi, d_F, nim*sizeof(float), hipMemcpyDeviceToHost);
	for (i=0; i<nim; ++i) {F[i] = (float)Fi[i] / scale;}
	// Free mem
	free(Fi);
	hipFree(d_im);
	hipFree(d_mumap);
	hipFree(d_F);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_z1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_z2);
	hipDeviceReset();
}


// 3D convolution (in Fourier)
void kernel_3Dconv_wrap_cuda(float* vol, int nz, int ny, int nx, float* H, int a, int b, int c) {
	int ID = 0;
	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// prepare the filter
	int nc = (ny / 2) + 1;
	int size_H = c * b * a;
	int size_vol = nz * ny * nx;
	int size_fft = nz * nc * nx;
	
	hipfftHandle plan_forward, plan_inverse;
	hipfftReal* dvol;
	hipfftComplex* dfft;
	float* dH;
	int status;

	// alloc mem GPU
	status = hipMalloc((void**)&dvol, size_vol * sizeof(hipfftReal));
	//printf("dvol %i\n", status);
	status = hipMalloc((void**)&dfft, size_fft * sizeof(hipfftComplex));
	//printf("dfft %i\n", status);
	status = hipMalloc((void**)&dH, size_H * sizeof(float));
	//printf("dH %i\n", status);
	
	// tranfert to GPU
	status = hipMemcpy(dvol, vol, size_vol * sizeof(hipfftReal), hipMemcpyHostToDevice);
	//printf("memcpy dvol %i\n", status);
	status = hipMemcpy(dH, H, size_H * sizeof(float), hipMemcpyHostToDevice);
	//printf("memcpy dH %i\n", status);
	
	// do fft
	status = hipfftPlan3d(&plan_forward, nx, ny, nz, HIPFFT_R2C);
	//printf("init plan %i\n", status);
	status = hipfftExecR2C(plan_forward, dvol, dfft);
	//printf("fft %i\n", status);
	
	// do 3D convolution
	int block_size, grid_size;
	dim3 threads, grid;
	block_size = 128;
	grid_size = (size_fft + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	vector_complex_x_real<<<grid, threads>>>(dfft, dH, size_fft);

	// get inverse transform
	hipfftPlan3d(&plan_inverse, nz, ny, nx, HIPFFT_C2R);
	hipfftExecC2R(plan_inverse, dfft, dvol);

	// Normalize values due to FFT theorem (1 / N)
	block_size = 128;
	grid_size = (size_vol + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	vector_real_x_cst<<<grid, threads>>>(dvol, 1 / float(size_vol), size_vol);

	// get back the volume
	hipMemcpy(vol, dvol, size_vol * sizeof(float), hipMemcpyDeviceToHost);
	
	// clean up
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_inverse);
	hipFree(dvol);
	hipFree(dH);
	hipFree(dfft);
	
	hipDeviceReset();
}


////////////////////////////////////////////////////////////////////////
// 3D-OPLEM
////////////////////////////////////////////////////////////////////////

// DDA ray-projector
__global__ void pet3D_OPLEM_DDA_V0(unsigned int* d_F, float* d_im,
								   unsigned short int* d_x1, unsigned short int* d_y1, unsigned short int* d_z1,
								   unsigned short int* d_x2, unsigned short int* d_y2, unsigned short int* d_z2,
								   int sublor_start, int sublor_stop, int nim3, int nim, int nsublor, float scale) {

	int length, n, diffx, diffy, diffz, step;
	float flength, x, y, z, lx, ly, lz, xinc, yinc, zinc, Qi;
	unsigned short int x1, y1, z1;
	int idx = blockIdx.x * blockDim.x + threadIdx.x + sublor_start;
	step = nim3*nim3;
	
	if (idx < sublor_stop) {
		Qi = 0.0f;
		//x1 = tex1Dfetch(tex_x1, idx);
		//y1 = tex1Dfetch(tex_y1, idx);
		//z1 = tex1Dfetch(tex_z1, idx);
		//diffx = tex1Dfetch(tex_x2, idx)-x1;
		//diffy = tex1Dfetch(tex_y2, idx)-y1;
		//diffz = tex1Dfetch(tex_z2, idx)-z1;
		x1 = d_x1[idx];
		y1 = d_y1[idx];
		z1 = d_z1[idx];
		diffx = d_x2[idx] - x1;
		diffy = d_y2[idx] - y1;
		diffz = d_z2[idx] - z1;
		lx = abs(diffx);
		ly = abs(diffy);
		lz = abs(diffz);
		length = ly;
		if (lx > length) {length = lx;}
		if (lz > length) {length = lz;}
		flength = 1.0f / (float)length;
		xinc = diffx * flength;
		yinc = diffy * flength;
		zinc = diffz * flength;
		x = x1;
		y = y1;
		z = z1;
		for (n=0; n<=length; ++n) {
			//Qi = Qi + tex1Dfetch(tex_im, (int)z * step + (int)y * wim + (int)x);
			Qi = Qi + d_im[(int)z * step + (int)y * nim3 + (int)x];
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}

		// compute F
		if (Qi==0.0f) {return;}
		Qi = 1 / Qi;
		x = x1;
		y = y1;
		z = z1;
		for (n=0; n<=length; ++n) {
			//atomicFloatAdd(&d_F[(int)z * step + (int)y * wim + (int)x], Qi);
			atomicAdd(&d_F[(int)z * step + (int)y * nim3 + (int)x], (unsigned int)(Qi*scale));
			x = x + xinc;
			y = y + yinc;
			z = z + zinc;
		}
	}
}

__global__ void toto(unsigned int* d_F, unsigned short int* d_x1) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	d_F[d_x1[idx]] = d_x1[idx];
}

// OPL-3D-OSEM algorithm with DDA-ELL
void kernel_pet3D_OPLEM_wrap_cuda_V0(unsigned short int* x1, int nx1, unsigned short int* y1, int ny1,
									 unsigned short int* z1, int nz1, unsigned short int* x2, int nx2,
									 unsigned short int* y2, int ny2, unsigned short int* z2, int nz2,
									 float* im, int nim1, int nim2, int nim3,
									 float* NM, int NM1, int NM2, int NM3, int Nsub, int ID){
	
	// Constant according Graphical card
	int mem_max = 800000000; // only 800 MB on 1 GB required
	float scale = 4000.0f;
	
	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// vars
	
	int block_size, grid_size, i;
	dim3 threads, grid;
	int nim = nim1 * nim2 * nim3;
	unsigned short int* d_x1;
	unsigned short int* d_x2;
	unsigned short int* d_y1;
	unsigned short int* d_y2;
	unsigned short int* d_z1;
	unsigned short int* d_z2;
	
	unsigned int mem_size_point;
	
	// memory managment 12 B / LORs and 8 B / Vox
	int max_sub, Nouter, mem_sub;
	mem_max -= (8 * nim);
	mem_sub = 12.0f * nx1 / float(Nsub);
	max_sub = int(mem_max / mem_sub);
	Nouter = (Nsub + max_sub - 1) / max_sub;

	printf("max_sub %i   Nouter %i\n", max_sub, Nouter);

	// device mem allocation
	float* d_im;
	unsigned int mem_size_im = nim * sizeof(float);
	hipMalloc((void**) &d_im, mem_size_im);
	unsigned int* d_F;
	unsigned int mem_size_F = nim * sizeof(int);
	hipMalloc((void**) &d_F, mem_size_F);
	unsigned int* F = (unsigned int*)malloc(mem_size_F);

	// Outer loop, avoid memory overflow
	int sub_start, sub_stop, nsub;
	int lor_start, lor_stop, nlor;
	int sublor_start, sublor_stop, nsublor;
	for (int iouter=0; iouter < Nouter; ++iouter) {
		printf("iouter %i\n", iouter);
		// split subset
		sub_start = int(float(Nsub) / Nouter * iouter + 0.5f);
		sub_stop = int(float(Nsub) / Nouter * (iouter+1) + 0.5f);
		nsub = sub_stop - sub_start;
		printf("   sub: %i to %i\n", sub_start, sub_stop);
		// load in memory subsets
		lor_start = int(float(nx1) / Nsub * sub_start + 0.5f);
		lor_stop = int(float(nx1) / Nsub * sub_stop + 0.5f);
		nlor = lor_stop - lor_start;
		printf("   lor: %i to %i\n", lor_start, lor_stop);
		mem_size_point = nlor * sizeof(short int);
		hipMalloc((void**) &d_x1, mem_size_point);
		hipMalloc((void**) &d_y1, mem_size_point);
		hipMalloc((void**) &d_z1, mem_size_point);
		hipMalloc((void**) &d_x2, mem_size_point);
		hipMalloc((void**) &d_y2, mem_size_point);
		hipMalloc((void**) &d_z2, mem_size_point);
		hipMemcpy(d_x1, &x1[lor_start], mem_size_point, hipMemcpyHostToDevice);
		hipMemcpy(d_y1, &y1[lor_start], mem_size_point, hipMemcpyHostToDevice);
		hipMemcpy(d_z1, &z1[lor_start], mem_size_point, hipMemcpyHostToDevice);
		hipMemcpy(d_x2, &x2[lor_start], mem_size_point, hipMemcpyHostToDevice);
		hipMemcpy(d_y2, &y2[lor_start], mem_size_point, hipMemcpyHostToDevice);
		hipMemcpy(d_z2, &z2[lor_start], mem_size_point, hipMemcpyHostToDevice);
		//hipBindTexture(NULL, tex_x1, d_x1, mem_size_point);
		//hipBindTexture(NULL, tex_y1, d_y1, mem_size_point);
		//hipBindTexture(NULL, tex_z1, d_z1, mem_size_point);
		//hipBindTexture(NULL, tex_x2, d_x2, mem_size_point);
		//hipBindTexture(NULL, tex_y2, d_y2, mem_size_point);
		//hipBindTexture(NULL, tex_z2, d_z2, mem_size_point);

		hipMemcpy(d_im, im, mem_size_im, hipMemcpyHostToDevice);
		// subset loop
		int c=0;
		for (int isub=0; isub < nsub; ++isub) {
			//isub=0;
			printf("   isub: %i\n", isub);
			sublor_start = int(float(nlor) / nsub * isub + 0.5f);
			sublor_stop = int(float(nlor) / nsub * (isub+1) + 0.5f);
			nsublor = sublor_stop - sublor_start;
			printf("      sublor: %i to %i\n", sublor_start, sublor_stop);

			/*
			// init F and load im to the GPU
			for (i=0; i<nim; ++i) {F[i] = 0;}
			hipMemcpy(d_F, F, mem_size_F, hipMemcpyHostToDevice);
			
			// kernel
			block_size = 256;
			grid_size = (nsublor + block_size - 1) / block_size; // CODE IS LIMITED TO < 16e6 lines
			threads.x = block_size;
			grid.x = grid_size;
			//pet3D_OPLEM_DDA_V0<<<grid, threads>>>(d_F, d_im, d_x1, d_y1, d_z1, d_x2, d_y2, d_z2,
			//									  sublor_start, sublor_stop, nim3, nim, nsublor, scale);
			toto<<<grid, threads>>>(d_F, d_x1);
			
			// get back F
			hipMemcpy(F, d_F, mem_size_F, hipMemcpyDeviceToHost);
			//hipMemcpy(im, d_im, mem_size_im, hipMemcpyDeviceToHost);
			// update volume
			scale = 1 / scale;
			int fmax=0;
			int imax=0;
			for (i=0; i<nim; ++i) {
				if (F[i] > fmax) {fmax = F[i];}
				//im[i] = im[i] * (float)F[i] * scale / NM[i];
				if (im[i] > imax) {imax = im[i];}
			}
			printf("      FMAX: %i\n", fmax);
			printf("      IMAX: %i\n", imax);
			// clean
			++c;
			if (c==2) {break;}
			*/

		} // isub
		// clean
		hipFree(d_x1);
		hipFree(d_y1);
		hipFree(d_z1);
		hipFree(d_x2);
		hipFree(d_y2);
		hipFree(d_z2);
		
	} // iouter
	free(F);
	hipFree(d_im);
	hipFree(d_F);
	hipDeviceReset();
}
