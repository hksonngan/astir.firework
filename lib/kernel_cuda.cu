#include "hip/hip_runtime.h"
#include "kernel_cuda.h"
#include <stdio.h>
#include <hipblas.h>
#include <sys/time.h>

// declare texture ref for 1D float texture
texture<float, 1, hipReadModeElementType> tex;

// kernel to update image in pet2D EMML algorithm
__global__ void pet2D_im_update(float* im, float* S, float* F, int npix) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < npix && F[idx] != 0.0f) {
		im[idx] = im[idx] / S[idx] * F[idx];
	}
}
// kernel to update Q value in pet2D EMML algorithm
__global__ void pet2D_Q_update(int* d_lorval, float* d_Q, int nval) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nval) {
		d_Q[idx] = (float)d_lorval[idx] / d_Q[idx];
	}
}
// kernel to raytrace line in SRM with DDA algorithm
__global__ void pet2D_SRM_DDA(float* d_SRM, int* d_X1, int* d_Y1, int* d_X2, int* d_Y2, int wx, int nx1, int width_image) {
	int length, n, x1, y1, x2, y2, diffx, diffy, LOR_ind;
	float flength, val, x, y, lx, ly, xinc, yinc;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nx1) {
		LOR_ind = idx * wx;
		x1 = d_X1[idx];
		y1 = d_Y1[idx];
		x2 = d_X2[idx];
		y2 = d_Y2[idx];
		diffx = x2-x1;
		diffy = y2-y1;
		lx = abs(diffx);
		ly = abs(diffy);
		length = ly;
		if (lx > length) {length = lx;}
		flength = (float)length;
		xinc = diffx / flength;
		yinc = diffy / flength;
		val = 1.0f / flength;
		x = x1 + 0.5f;
		y = y1 + 0.5f;
		for (n=0; n<=length; ++n) {
			d_SRM[LOR_ind + (int)y * width_image + (int)x] = val;
			x = x + xinc;
			y = y + yinc;
		}
	}
}

__global__ void matrix_ell_spmv(float* d_vals, int* d_cols, float* d_res, int niv, int njv) {
	int j, ind, vcol;
	float sum;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < niv) {
		ind = idx * njv;
		vcol = d_cols[ind];
		j = 0;
		sum = 0.0f;
		while (vcol != -1) {
			sum += (d_vals[ind+j] * tex1D(tex, vcol));
			++j;
			vcol = d_cols[ind+j];
		}
		d_res[idx] = sum;
	}
}

void kernel_pet2D_EMML_wrap_cuda(float* SRM, int nlor, int npix, float* im, int npixim, int* LOR_val, int nval, float* S, int ns, int maxit) {
	// select a GPU
	hipSetDevice(0);
	// init cublas
	hipblasStatus_t status;
	status = cublasInit();
	// allocate device memory for SRM, im, Q and F
	int size_SRM = nlor * npix;
	float* d_SRM;
	float* d_im;
	float* d_Q;
	float* d_F;
	float* d_S;
	int* d_lorval;
	status = cublasAlloc(size_SRM, sizeof(float), (void**)&d_SRM);
	status = cublasAlloc(npixim, sizeof(float), (void**)&d_im);
	status = cublasAlloc(nlor, sizeof(float), (void**)&d_Q);
	status = cublasAlloc(nlor, sizeof(float), (void**)&d_F);
	status = cublasAlloc(ns, sizeof(float), (void**)&d_S);
	status = cublasAlloc(nval, sizeof(int), (void**)&d_lorval);
	// load SRM, SM, LOR_val and im to the device
	status = hipblasSetVector(size_SRM, sizeof(float), SRM, 1, d_SRM, 1);
	status = hipblasSetVector(npixim, sizeof(float), im, 1, d_im, 1);
	status = hipblasSetVector(ns, sizeof(float), S, 1, d_S, 1);
	status = hipblasSetVector(nval, sizeof(int), LOR_val, 1, d_lorval, 1);
	if (status != 0) {exit(0);}
	int ite, block_size1, grid_size1, block_size2, grid_size2;
	block_size1 = 256;
	grid_size1 = (nlor + block_size1 - 1) / block_size1;
	block_size2 = 64;
	grid_size2 = (npix + block_size2 - 1) / block_size2;
	dim3 threads1(block_size1);
	dim3 grid1(grid_size1);
	dim3 threads2(block_size2);
	dim3 grid2(grid_size2);
	for (ite=0; ite<maxit; ++ite) {
		// compute Q
		hipblasSgemv('t', npix, nlor, 1.0, d_SRM, npix, d_im, 1, 0.0, d_Q, 1);
		pet2D_Q_update<<< grid1, threads1 >>>(d_lorval, d_Q, nval);
		// compute f = sum{SRMi / qi} for each i LOR
		hipblasSgemv('n', npix, nlor, 1.0, d_SRM, npix, d_Q, 1, 0.0, d_F, 1);
		// update image
		pet2D_im_update<<< grid2, threads2 >>>(d_im, d_S, d_F, npix);
	}
	// get results
	status = cublasGetError();
	status = hipblasGetVector(npix, sizeof(float), d_im, 1, im, 1);
	// free memory
	status = cublasFree(d_SRM);
	status = cublasFree(d_im);
	status = cublasFree(d_Q);
	status = cublasFree(d_F);
	status = cublasFree(d_S);
	// prepare to quit
	status = cublasShutdown();
}

void kernel_pet2D_SRM_DDA_wrap_cuda(float* SRM, int wy, int wx, int* X1, int nx1, int* Y1, int ny1, int* X2, int nx2, int* Y2, int ny2, int width_image) {
	// select a GPU
	hipSetDevice(0);
	// some vars
	int size_SRM = wy * wx;
	unsigned int mem_size_SRM = sizeof(float) * size_SRM;
	unsigned int mem_size_point = sizeof(int) * nx1;
	// alloacte device memory for SRM, x1, y1, x2, and y2
	float* d_SRM;
	int* d_X1;
	int* d_Y1;
	int* d_X2;
	int* d_Y2;
	hipMalloc((void**) &d_SRM, mem_size_SRM);
	hipMemset(d_SRM, 0.0f, mem_size_SRM);
	hipMalloc((void**) &d_X1, mem_size_point);
	hipMalloc((void**) &d_Y1, mem_size_point);
	hipMalloc((void**) &d_X2, mem_size_point);
	hipMalloc((void**) &d_Y2, mem_size_point);
	// copy host memory to device
	hipMemcpy(d_X1, X1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_Y1, Y1, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_X2, X2, mem_size_point, hipMemcpyHostToDevice);
	hipMemcpy(d_Y2, Y2, mem_size_point, hipMemcpyHostToDevice);
	// setup execution parameters
	int block_size, grid_size;
	block_size = 256;
	grid_size = (nx1 + block_size - 1) / block_size;
	dim3 threads(block_size);
	dim3 grid(grid_size);

	//timeval start, end;
	//double t1, t2, diff;
	//gettimeofday(&start, NULL);
	//t1 = start.tv_sec + start.tv_usec / 1000000.0;
	// DDA kernel
	pet2D_SRM_DDA<<< grid, threads >>>(d_SRM, d_X1, d_Y1, d_X2, d_Y2, wx, nx1, width_image);
	hipDeviceSynchronize();
	// get back results to the host
	hipMemcpy(SRM, d_SRM, mem_size_SRM, hipMemcpyDeviceToHost);
	//gettimeofday(&end, NULL);
	//t2 = end.tv_sec + end.tv_usec / 1000000.0;
	//diff = t2 - t1;
	//printf("time %f s\n", diff);
	// clean up memory
	hipFree(d_SRM);
	hipFree(d_X1);
	hipFree(d_Y1);
	hipFree(d_X2);
	hipFree(d_Y2);
}

void kernel_matrix_ell_spmv_wrap_cuda(float* vals, int niv, int njv, int* cols, int nic, int njc, float* y, int ny, float* res, int nres) {
	// select a GPU
	hipSetDevice(0);
	// some vars
	int size_data = niv * njv;
	unsigned int mem_size_dataf = sizeof(float) * size_data;
	unsigned int mem_size_y = sizeof(float) * ny;
	unsigned int mem_size_res = sizeof(float) * nres;
	unsigned int mem_size_datai = sizeof(int) * size_data;
	// alloacte device memory
	float* d_vals;
	float* d_res;
	//float* d_y;
	int* d_cols;
	hipMalloc((void**) &d_vals, mem_size_dataf);
	hipMalloc((void**) &d_res, mem_size_res);
	//hipMalloc((void**) &d_y, mem_size_y);
	hipMalloc((void**) &d_cols, mem_size_datai);
	// copy host memory to device
	hipMemcpy(d_vals, vals, mem_size_dataf, hipMemcpyHostToDevice);
	//hipMemcpy(d_y, y, mem_size_res, hipMemcpyHostToDevice);
	hipMemcpy(d_cols, cols, mem_size_datai, hipMemcpyHostToDevice);
	// prepare texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    hipMallocArray( &cu_array, &channelDesc, ny, 1 ); 
    hipMemcpyToArray(cu_array, 0, 0, y, mem_size_y, hipMemcpyHostToDevice);
	tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;
    hipBindTextureToArray(tex, cu_array, channelDesc);

	// setup execution parameters
	int block_size, grid_size;
	block_size = 256;
	grid_size = (niv + block_size - 1) / block_size;
	dim3 threads(block_size);
	dim3 grid(grid_size);
	timeval start, end;
	double t1, t2, diff;
	gettimeofday(&start, NULL);
	t1 = start.tv_sec + start.tv_usec / 1000000.0;
	// spmv kernel
	matrix_ell_spmv<<< grid, threads >>>(d_vals, d_cols, d_res, niv, njv);
	hipDeviceSynchronize();
	// get back results to the host
	hipMemcpy(res, d_res, mem_size_res, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	t2 = end.tv_sec + end.tv_usec / 1000000.0;
	diff = t2 - t1;
	printf("kernel time %f s\n", diff);
	// clean up memory
	hipFree(d_vals);
	hipFree(d_cols);
	//hipFree(d_y);
	hipFree(d_res);
}
