#include "hip/hip_runtime.h"
#include "kernel_cuda.h"
#include <stdio.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <hip/hip_math_constants.h>

// Perform a multiplication between a complex and a real vectors
__global__ void vector_complex_x_real(hipfftComplex* dcpx, float* dr, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float r, j, h;
	if (idx < n) {
		r = dcpx[idx].x;
		j = dcpx[idx].y;
		h = dr[idx];
		r = r * h;
		j = j * h;
		dcpx[idx].x = r;
		dcpx[idx].y = j;
	}
}

// Perform a mulitplication between a real vectors and an alpha value
__global__ void vector_real_x_cst(float* dr, float alpha, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float val;
	if (idx < n) {
		val = dr[idx];
		val = val * alpha;
		dr[idx] = val;
	}
}

// 3D convolution (in Fourier)
void kernel_3D_conv_wrap_cuda(float* vol, int nz, int ny, int nx, float* H, int a, int b, int c) {
	int ID = 0;
	// select a GPU
	if (ID != -1){hipSetDevice(ID);}
	// prepare the filter
	int nc = (ny / 2) + 1;
	int size_H = c * b * a;
	int size_vol = nz * ny * nx;
	int size_fft = nz * nc * nx;
	
	hipfftHandle plan_forward, plan_inverse;
	hipfftReal* dvol;
	hipfftComplex* dfft;
	float* dH;

	// alloc mem GPU
	hipMalloc((void**)&dvol, size_vol * sizeof(hipfftReal));
	//printf("dvol %i\n", status);
	hipMalloc((void**)&dfft, size_fft * sizeof(hipfftComplex));
	//printf("dfft %i\n", status);
	hipMalloc((void**)&dH, size_H * sizeof(float));
	//printf("dH %i\n", status);
	
	// tranfert to GPU
	hipMemcpy(dvol, vol, size_vol * sizeof(hipfftReal), hipMemcpyHostToDevice);
	//printf("memcpy dvol %i\n", status);
	hipMemcpy(dH, H, size_H * sizeof(float), hipMemcpyHostToDevice);
	//printf("memcpy dH %i\n", status);
	
	// do fft
	hipfftPlan3d(&plan_forward, nx, ny, nz, HIPFFT_R2C);
	//printf("init plan %i\n", status);
	hipfftExecR2C(plan_forward, dvol, dfft);
	//printf("fft %i\n", status);
	
	// do 3D convolution
	int block_size, grid_size;
	dim3 threads, grid;
	block_size = 128;
	grid_size = (size_fft + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	vector_complex_x_real<<<grid, threads>>>(dfft, dH, size_fft);

	// get inverse transform
	hipfftPlan3d(&plan_inverse, nz, ny, nx, HIPFFT_C2R);
	hipfftExecC2R(plan_inverse, dfft, dvol);

	// Normalize values due to FFT theorem (1 / N)
	block_size = 128;
	grid_size = (size_vol + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;
	vector_real_x_cst<<<grid, threads>>>(dvol, 1 / float(size_vol), size_vol);

	// get back the volume
	hipMemcpy(vol, dvol, size_vol * sizeof(float), hipMemcpyDeviceToHost);
	
	// clean up
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_inverse);
	hipFree(dvol);
	hipFree(dH);
	hipFree(dfft);
	
	hipDeviceReset();
}

