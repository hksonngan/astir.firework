#include "hip/hip_runtime.h"
// This file is part of FIREwork
// 
// FIREwork is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// FIREwire is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with FIREwork.  If not, see <http://www.gnu.org/licenses/>.
//
// FIREwork Copyright (C) 2008 - 2011 Julien Bert 

#include "mc_cuda.h"
#include "mc_cuda_cst.cu"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <hip/hip_math_constants.h>

/***********************************************************
 * Utils
 ***********************************************************/
__constant__ const float pi = 3.14159265358979323846;
__constant__ const float twopi = 2*pi;
texture<float, 1, hipReadModeElementType> tex_vol;

// Stack of gamma particles, format data is defined as SoA
struct StackGamma{
	float* E;
	float* dx;
	float* dy;
	float* dz;
	float* px;
	float* py;
	float* pz;
	int* seed;
	int* ct_eff;
	int* ct_Cpt;
	int* ct_PE;
	unsigned char* live;
	unsigned char* in;
	unsigned int size;
}; //

// Given by Hector doesn't work properly
__device__ float park_miller(long unsigned int *seed) {
	long unsigned int hi, lo;
	int const a = 16807;
	int const m = 2147483647;
	float const recm = __fdividef(1.0f, m);

	lo = a * (*seed & 0xFFFF);
	hi = a * (*seed >> 16);
	lo += (hi & 0x7FFF) << 16;
	lo += (hi >> 15);
	if (lo > 0x7FFFFFFF) {lo -= 0x7FFFFFFF;}
	*seed = (long)lo;

	return (*seed)*recm;
}

// Park-Miller from C numerical book
__device__ float park_miller_jb(int *seed) {
	int const a = 16807;
	int const m = 2147483647;
	int const iq = 127773;
	int const ir = 2836;
	int const mask = 123459876;
	float const recm = __fdividef(1.0f, m);
	int k;
	float ans;
	*seed ^= mask;
	k = (*seed)/iq;
	*seed = a * (*seed-k*iq) - ir*k;
	if (*seed < 0) {*seed += m;}
	ans = recm * (*seed);
	*seed ^= mask;
	return ans;
}

// Hamilton multiplication (quaternion)
__device__ float4 quat_mul(float4 p, float4 q) {
	return make_float4(
		   p.w*q.x + p.x*q.w + p.y*q.z - p.z*q.y,    // x
		   p.w*q.y + p.y*q.w + p.z*q.x - p.x*q.z,    // y
		   p.w*q.z + p.z*q.w + p.x*q.y - p.y*q.x,    // z
		   p.w*q.w - p.x*q.x - p.y*q.y - p.z*q.z);   // w
}

// Create quaternion for axis angle rotation
__device__ float4 quat_axis(float4 n, float theta) {
	theta /= 2.0f;
	float stheta = __sinf(theta);
	return make_float4(n.x*stheta, n.y*stheta, n.z*stheta, __cosf(theta));
}

// Conjugate quaternion
__device__ float4 quat_conj(float4 p) {
	return make_float4(-p.x, -p.y, -p.z, p.w);
}

// Normalize quaternion
__device__ float4 quat_norm(float4 p) {
	float norm = __fdividef(1.0f, __powf(p.w*p.w+p.x*p.x+p.y*p.y+p.z*p.z, 0.5f));
	return make_float4(p.x*norm, p.y*norm, p.z*norm, p.w*norm);
}

// Cross product
__device__ float4 quat_crossprod(float4 u, float4 v){
	return make_float4(u.y*v.z-u.z*v.y, u.z*v.x-u.x*v.z, u.x*v.y-u.y*v.x, 0.0f);
}

/***********************************************************
 * Physics
 ***********************************************************/
// Compton Cross Section Per Atom
__device__ float Compton_CSPA(float E, float Z) {
	float CrossSection = 0.0;
	if (Z<0.9999f || E < 1e-4f) {return CrossSection;}

	float p1Z = Z*(2.7965e-23f + 1.9756e-27f*Z + -3.9178e-29f*Z*Z);
	float p2Z = Z*(-1.8300e-23f + -1.0205e-24f*Z + 6.8241e-27f*Z*Z);
	float p3Z = Z*(6.7527e-22f + -7.3913e-24f*Z + 6.0480e-27f*Z*Z);
	float p4Z = Z*(-1.9798e-21f + 2.7079e-24f*Z + 3.0274e-26f*Z*Z);
	float T0 = (Z < 1.5f)? 40.0e-3f : 15.0e-3f;
	float d1, d2, d3, d4, d5;

	d1 = __fdividef(fmaxf(E, T0), 0.510998910f); // X
	CrossSection = __fdividef(p1Z*__logf(1.0f+2.0f*d1), d1) + __fdividef(p2Z + p3Z*d1 + p4Z*d1*d1, 1.0f + 20.0f*d1 + 230.0f*d1*d1 + 440.0f*d1*d1*d1);

	if (E < T0) {
		d1 = __fdividef(T0+1.0e-3f, 0.510998910f); // X
		d2 = __fdividef(p1Z*__logf(1.0f+2.0f*d1), d1) + __fdividef(p2Z + p3Z*d1 + p4Z*d1*d1, 1.0f + 20.0f*d1 + 230.0f*d1*d1 + 440.0f*d1*d1*d1); // sigma
		d3 = __fdividef(-T0 * (d2 - CrossSection), CrossSection*1.0e-3f); // c1
		d4 = (Z > 1.5f)? 0.375f-0.0556f*__logf(Z) : 0.15f; // c2
		d5 = __logf(__fdividef(E, T0)); // y
		CrossSection *= __expf(-d5 * (d3 + d4*d5));
	}

	return CrossSection;
}

// Compton Scatter (Klein-Nishina)
__device__ float Compton_scatter(StackGamma stackgamma, unsigned int id) {
	float E = stackgamma.E[id];
	int seed = stackgamma.seed[id];
	float E0 = __fdividef(E, 0.510998910f);

	float epszero = __fdividef(1.0f, (1.0f + 2.0f * E0));
	float eps02 = epszero*epszero;
	float a1 = -__logf(epszero);
	float a2 = __fdividef(a1, (a1 + 0.5f*(1.0f-eps02)));

	float greject, onecost, eps, eps2;
	do {
		if (a2 > park_miller_jb(&seed)) {
			eps = __expf(-a1 * park_miller_jb(&seed));
			eps2 = eps*eps;
		} else {
			eps2 = eps02 + (1.0f - eps02) * park_miller_jb(&seed);
			eps = sqrt(eps2);
		}
		onecost = __fdividef(1.0f - eps, eps * E0);
		greject = 1.0f - eps * onecost * __fdividef(2.0f - onecost, 1.0f + eps2);
	} while (greject < park_miller_jb(&seed));

	E *= eps;
	stackgamma.seed[id] = seed;
	stackgamma.E[id] = E;
	if (E <= 1.0e-6f) {
		stackgamma.live[id] = 0;
		return 0.0f;
	}
	
	return acos(1.0f - onecost);
}

// PhotoElectric Cross Section Per Atom, use Sandia data and load 21,236 Bytes on constant memory.
__device__ float PhotoElec_CSPA(float E, int Z) {
	float Emin = fmax(fIonizationPotentials[Z]*1e-6f, 0.01e-3f); // from Sandia, the same for all Z
	if (E < Emin) {return 0.0f;}
	
	int start = fCumulIntervals[Z];
	int stop = start + fNbOfIntervals[Z] - 1.0f;
	int pos;
	for (pos=stop; pos>start; --pos) {
		if (E < fSandiaTable[pos][0]*1.0e-3f) {break;}
	}
	float AoverAvo = 103.642688246e-10f * __fdividef((float)Z, fZtoAratio[Z]);
	float rE = __fdividef(1.0f, E);
	float rE2 = rE*rE;

	return rE * fSandiaTable[pos][1] * AoverAvo * 0.160217648e-22f
		+ rE2 * fSandiaTable[pos][2] * AoverAvo * 0.160217648e-25f
		+ rE * rE2 * fSandiaTable[pos][3] * AoverAvo * 0.160217648e-28f
		+ rE2 * rE2 * fSandiaTable[pos][4] * AoverAvo * 0.160217648e-31f;
}

__device__ float Compton_mu_Water(float E) {
	// H2O
	return (2*Compton_CSPA(E, 1) + Compton_CSPA(E, 8)) * 3.342796664e+19f; // Avogadro*H2O_density / (2*a_H+a_O)
}
__device__ float Compton_mu_Plastic(float E) {
	// 5C8H2O
	return (5*Compton_CSPA(E, 6) + 8*Compton_CSPA(E, 1) + 2*Compton_CSPA(E, 8)) * 7.096901340e17f;
}
__device__ float Compton_mu_Al(float E) {
	// Al
	return Compton_CSPA(E, 13) * 6.024030465e+19f; // Avogadro*Al_density/a_Al
}
__device__ float Compton_mu_Air(float E) {
	// O N Ar C
	return (0.231781f*Compton_CSPA(E, 8) + 0.755268f*Compton_CSPA(E, 7)
			+ 0.012827f*Compton_CSPA(E, 18) + 0.000124f*Compton_CSPA(E, 6)) * 5.247706935e17f;
}
__device__ float Compton_mu_Body(float E) {
	// H O
	return (0.112f*Compton_CSPA(E, 1) + 0.888f*Compton_CSPA(E, 8)) * 4.205077389e18f;
}
__device__ float Compton_mu_Lung(float E) {
	// H C N O Na P S Cl K
	return (0.103f*Compton_CSPA(E, 1)+ 0.105f*Compton_CSPA(E, 6) + 0.031f*Compton_CSPA(E, 7)
			+ 0.749f*Compton_CSPA(E, 8) + 0.002f*Compton_CSPA(E, 11) + 0.002f*Compton_CSPA(E, 15)
			+ 0.003f*Compton_CSPA(E, 16) + 0.003f*Compton_CSPA(E, 17) + 0.002f*Compton_CSPA(E, 19)) * 1.232299227e18f;
}
__device__ float Compton_mu_RibBone(float E) {
	// H C N O Na Mg P S Ca
	return (0.034f*Compton_CSPA(E, 1) + 0.155f*Compton_CSPA(E, 6) + 0.042f*Compton_CSPA(E, 7)
			+ 0.435f*Compton_CSPA(E, 8) + 0.001f*Compton_CSPA(E, 11) + 0.002f*Compton_CSPA(E, 12)
			+ 0.103f*Compton_CSPA(E, 15) + 0.003f*Compton_CSPA(E, 16) + 0.225f*Compton_CSPA(E, 20)) * 5.299038816e18f;
}
__device__ float Compton_mu_SpineBone(float E) {
	// H C N O Na Mg P S Cl K Ca
	return (0.063f*Compton_CSPA(E, 1) + 0.261f*Compton_CSPA(E, 6) + 0.039f*Compton_CSPA(E, 7)
			+ 0.436f*Compton_CSPA(E, 8) + 0.001f*Compton_CSPA(E, 11) + 0.001f*Compton_CSPA(E, 12)
			+ 0.061f*Compton_CSPA(E, 15) + 0.003f*Compton_CSPA(E, 16) + 0.001f*Compton_CSPA(E, 17)
			+ 0.001f*Compton_CSPA(E, 19) + 0.133f*Compton_CSPA(E, 20)) * 4.709337384e18f;
}
__device__ float Compton_mu_Heart(float E) {
	// H C N O Na P S Cl K
	return (0.104f*Compton_CSPA(E, 1) + 0.139f*Compton_CSPA(E, 6) + 0.029f*Compton_CSPA(E, 7)
			+ 0.718f*Compton_CSPA(E, 8) + 0.001f*Compton_CSPA(E, 11) + 0.002f*Compton_CSPA(E, 15)
			+ 0.002f*Compton_CSPA(E, 16) + 0.002f*Compton_CSPA(E, 17) + 0.003f*Compton_CSPA(E, 19)) * 4.514679219e18f;
}
__device__ float Compton_mu_Breast(float E) {
	// H C N O Na P S Cl
	return (0.106f*Compton_CSPA(E, 1) + 0.332f*Compton_CSPA(E, 6) + 0.03f*Compton_CSPA(E, 7)
			+ 0.527f*Compton_CSPA(E, 8) + 0.001f*Compton_CSPA(E, 11) + 0.001f*Compton_CSPA(E, 15)
			+ 0.002f*Compton_CSPA(E, 16) + 0.001f*Compton_CSPA(E, 17)) * 4.688916436e18f;
}

__device__ float PhotoElec_mu_Water(float E) {
	// H2O
	return (2*PhotoElec_CSPA(E, 1) + PhotoElec_CSPA(E, 8)) * 3.342796664e+19f; // Avogadro*H2O_density / (2*a_H+a_O)
}
__device__ float PhotoElec_mu_Plastic(float E) {
	// 5C8H2O
	return (5*PhotoElec_CSPA(E, 6) + 8*PhotoElec_CSPA(E, 1) + 2*PhotoElec_CSPA(E, 8)) * 7.096901340e17f;
}
__device__ float PhotoElec_mu_Al(float E) {
	// Al
	return PhotoElec_CSPA(E, 13) * 6.024030465e+19f; // Avogadro*Al_density/a_Al
}
__device__ float PhotoElec_mu_Air(float E) {
	// O N Ar C
	return (0.231781f*PhotoElec_CSPA(E, 8) + 0.755268f*PhotoElec_CSPA(E, 7)
			+ 0.012827f*PhotoElec_CSPA(E, 18) + 0.000124f*PhotoElec_CSPA(E, 6)) * 5.247706935e17f;
}
__device__ float PhotoElec_mu_Body(float E) {
	// H O
	return (0.112f*PhotoElec_CSPA(E, 1) + 0.888f*PhotoElec_CSPA(E, 8)) * 4.205077389e18f;
}
__device__ float PhotoElec_mu_Lung(float E) {
	// H C N O Na P S Cl K
	return (0.103f*PhotoElec_CSPA(E, 1)+ 0.105f*PhotoElec_CSPA(E, 6) + 0.031f*PhotoElec_CSPA(E, 7)
			+ 0.749f*PhotoElec_CSPA(E, 8) + 0.002f*PhotoElec_CSPA(E, 11) + 0.002f*PhotoElec_CSPA(E, 15)
			+ 0.003f*PhotoElec_CSPA(E, 16) + 0.003f*PhotoElec_CSPA(E, 17) + 0.002f*PhotoElec_CSPA(E, 19)) * 1.232299227e18f;
}
__device__ float PhotoElec_mu_RibBone(float E) {
	// H C N O Na Mg P S Ca
	return (0.034f*PhotoElec_CSPA(E, 1) + 0.155f*PhotoElec_CSPA(E, 6) + 0.042f*PhotoElec_CSPA(E, 7)
			+ 0.435f*PhotoElec_CSPA(E, 8) + 0.001f*PhotoElec_CSPA(E, 11) + 0.002f*PhotoElec_CSPA(E, 12)
			+ 0.103f*PhotoElec_CSPA(E, 15) + 0.003f*PhotoElec_CSPA(E, 16) + 0.225f*PhotoElec_CSPA(E, 20)) * 5.299038816e18f;
}
__device__ float PhotoElec_mu_SpineBone(float E) {
	// H C N O Na Mg P S Cl K Ca
	return (0.063f*PhotoElec_CSPA(E, 1) + 0.261f*PhotoElec_CSPA(E, 6) + 0.039f*PhotoElec_CSPA(E, 7)
			+ 0.436f*PhotoElec_CSPA(E, 8) + 0.001f*PhotoElec_CSPA(E, 11) + 0.001f*PhotoElec_CSPA(E, 12)
			+ 0.061f*PhotoElec_CSPA(E, 15) + 0.003f*PhotoElec_CSPA(E, 16) + 0.001f*PhotoElec_CSPA(E, 17)
			+ 0.001f*PhotoElec_CSPA(E, 19) + 0.133f*PhotoElec_CSPA(E, 20)) * 4.709337384e18f;
}
__device__ float PhotoElec_mu_Heart(float E) {
	// H C N O Na P S Cl K
	return (0.104f*PhotoElec_CSPA(E, 1) + 0.139f*PhotoElec_CSPA(E, 6) + 0.029f*PhotoElec_CSPA(E, 7)
			+ 0.718f*PhotoElec_CSPA(E, 8) + 0.001f*PhotoElec_CSPA(E, 11) + 0.002f*PhotoElec_CSPA(E, 15)
			+ 0.002f*PhotoElec_CSPA(E, 16) + 0.002f*PhotoElec_CSPA(E, 17) + 0.003f*PhotoElec_CSPA(E, 19)) * 4.514679219e18f;
}
__device__ float PhotoElec_mu_Breast(float E) {
	// H C N O Na P S Cl
	return (0.106f*PhotoElec_CSPA(E, 1) + 0.332f*PhotoElec_CSPA(E, 6) + 0.03f*PhotoElec_CSPA(E, 7)
			+ 0.527f*PhotoElec_CSPA(E, 8) + 0.001f*PhotoElec_CSPA(E, 11) + 0.001f*PhotoElec_CSPA(E, 15)
			+ 0.002f*PhotoElec_CSPA(E, 16) + 0.001f*PhotoElec_CSPA(E, 17)) * 4.688916436e18f;
}

// return attenuation according materials 
__device__ float att_from_mat(int mat, float E) {
	switch (mat) {
	case 0:     return Compton_mu_Air(E) + PhotoElec_mu_Air(E);
	case 1:     return Compton_mu_Body(E) + PhotoElec_mu_Body(E);
	case 2:     return Compton_mu_Lung(E) + PhotoElec_mu_Lung(E);
	case 3:     return Compton_mu_Breast(E) + PhotoElec_mu_Breast(E);
	case 4:     return Compton_mu_Heart(E) + PhotoElec_mu_Heart(E);
	case 5:     return Compton_mu_SpineBone(E) + PhotoElec_mu_SpineBone(E);
	case 6:     return Compton_mu_RibBone(E) + PhotoElec_mu_RibBone(E);
	case 98:    return Compton_mu_Plastic(E) + PhotoElec_mu_Plastic(E);
	case 99:	return Compton_mu_Water(E) + PhotoElec_mu_Water(E);
	case 100:	return Compton_mu_Al(E) + PhotoElec_mu_Al(E);
	}
	return 0.0f;
}


// Kernel interactions
__global__ void kernel_interactions(StackGamma stackgamma, float* ddose, int3 dimvol) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float theta, phi, dx, dy, dz, oldE, depdose;
	float Compton_CS, PhotoElec_CS, tot_CS, effect;
	int px, py, pz, jump, mat;
	int seed;
	if (id < stackgamma.size) {
		if (stackgamma.in[id] == 0) {return;} // if the particle is outside 
		
		seed = stackgamma.seed[id];
		dx = stackgamma.dx[id];
		dy = stackgamma.dy[id];
		dz = stackgamma.dz[id];
		px = int(stackgamma.px[id]);
		py = int(stackgamma.py[id]);
		pz = int(stackgamma.pz[id]);
		oldE = stackgamma.E[id];
		jump = dimvol.x * dimvol.y;
		//mat = int(dvol[pz*jump + py*dimvol.x + px]);
		mat = tex1Dfetch(tex_vol, pz*jump + py*dimvol.x + px);

		switch (mat) {
		case 0:     Compton_CS = Compton_mu_Air(oldE);       PhotoElec_CS = PhotoElec_mu_Air(oldE); break;
		case 1:     Compton_CS = Compton_mu_Body(oldE);      PhotoElec_CS = PhotoElec_mu_Body(oldE); break;
		case 2:     Compton_CS = Compton_mu_Lung(oldE);      PhotoElec_CS = PhotoElec_mu_Lung(oldE); break;
		case 3:     Compton_CS = Compton_mu_Breast(oldE);    PhotoElec_CS = PhotoElec_mu_Breast(oldE); break;
		case 4:     Compton_CS = Compton_mu_Heart(oldE);     PhotoElec_CS = PhotoElec_mu_Heart(oldE); break;
		case 5:     Compton_CS = Compton_mu_SpineBone(oldE); PhotoElec_CS = PhotoElec_mu_SpineBone(oldE); break;
		case 6:     Compton_CS = Compton_mu_RibBone(oldE);   PhotoElec_CS = PhotoElec_mu_RibBone(oldE); break;
		case 98:    Compton_CS = Compton_mu_Plastic(oldE);   PhotoElec_CS = PhotoElec_mu_Plastic(oldE); break;
		case 99:	Compton_CS = Compton_mu_Water(oldE);     PhotoElec_CS = PhotoElec_mu_Water(oldE); break;
		case 100:	Compton_CS = Compton_mu_Al(oldE);        PhotoElec_CS = PhotoElec_mu_Al(oldE); break;
		}

		// Select effect
		tot_CS = Compton_CS + PhotoElec_CS;
		PhotoElec_CS = __fdividef(PhotoElec_CS, tot_CS);
		Compton_CS = 1.0f;
		effect = park_miller_jb(&seed);

		if (effect <= PhotoElec_CS) {
			// PhotoElectric effect
			depdose = oldE;
			stackgamma.live[id] = 0;
			theta = 0.0f;
			phi = 0.0f;
			++stackgamma.ct_eff[id];
			++stackgamma.ct_PE[id];
		}
		if (effect > PhotoElec_CS && effect <= Compton_CS) {
			// Compton scattering
			theta = Compton_scatter(stackgamma, id);
			phi = park_miller_jb(&seed) * 2 * twopi;
			// !!!!! WARNING: should be 2*pi instead of 4*pi, it is to fix a pb with ParkMiller
			//                only uniform in half range ?! so the range must be twice.
			depdose = oldE - stackgamma.E[id];
			++stackgamma.ct_eff[id];
			++stackgamma.ct_Cpt[id];
		}

		// Dose depot
		ddose[pz*jump + py*dimvol.x + px] += depdose;
		// !!!!! WARNING: Atomic function is required (w/ ddose in uint)

		//*****************************************************
		// Apply new direction to the particle (use quaternion)
		//
		// create quaternion from particle and normalize it
		float4 p = make_float4(dx, dy, dz, 0.0f);
		p = quat_norm(p);
		// select best axis to compute the rotation axis
		float4 a = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		if (dx<dy) {a.x=1.0f;} // choose x axis
		else {a.y=1.0f;}       // choose y axis
		// create virtual axis given by p^a
		a = quat_crossprod(p, a);
		a = quat_norm(a);
		// build rotation around p axis with phi (in order to rotate the next rotation axis a)
		float4 r = quat_axis(p, phi);
		// do rotation of a = rar*
		a = quat_mul(a, quat_conj(r)); // a = ar*
		a = quat_mul(r, a);            // a = ra
		// build rotation around a axis with theta (thus rotate p)
		r = quat_axis(a, theta);
		// do final rotation of p = rpr*
		p = quat_mul(p, quat_conj(r));
		p = quat_mul(r, p);
		// assign new values
		stackgamma.dx[id] = p.x;
		stackgamma.dy[id] = p.y;
		stackgamma.dz[id] = p.z;
		stackgamma.seed[id] = seed;
	}
}


/***********************************************************
 * Managment
 ***********************************************************/
__global__ void kernel_particle_rnd(StackGamma stackgamma, int3 dimvol) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x)+threadIdx.x;
	float phi, theta;
	int seed = stackgamma.seed[id];
	// warmpup to diverge
	park_miller_jb(&seed);
	park_miller_jb(&seed);
	park_miller_jb(&seed);
	if (id < stackgamma.size) {
		// position
		stackgamma.px[id] = park_miller_jb(&seed)*dimvol.x;
		stackgamma.py[id] = park_miller_jb(&seed)*dimvol.y;
		stackgamma.pz[id] = park_miller_jb(&seed)*dimvol.z;
		// direction
		phi = park_miller_jb(&seed) * twopi;
		theta = park_miller_jb(&seed) * pi - 0.5*pi;
		stackgamma.dx[id] = __cosf(theta) * __cosf(phi);
		stackgamma.dy[id] = __cosf(theta) * __sinf(phi);
		stackgamma.dz[id] = __sinf(theta);
		// enable particles
		stackgamma.live[id] = 1;
		stackgamma.in[id] = 1;
		stackgamma.seed[id] = seed;
	}
}

__global__ void kernel_particle_gun(StackGamma stackgamma, int3 dimvol,
									float posx, float posy, float posz,
									float dx, float dy, float dz, float E) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (id < stackgamma.size) {
		if (stackgamma.in[id]==0 || stackgamma.live[id]==0) { 
			stackgamma.E[id] = E;
			stackgamma.px[id] = posx;
			stackgamma.py[id] = posy;
			stackgamma.pz[id] = posz;
			stackgamma.dx[id] = dx;
			stackgamma.dy[id] = dy;
			stackgamma.dz[id] = dz;
			stackgamma.live[id] = 1;
			stackgamma.in[id] = 1;
			stackgamma.ct_eff[id] = 0;
			stackgamma.ct_Cpt[id] = 0;
			stackgamma.ct_PE[id] = 0;
		}
	}
}

__global__ void kernel_particle_largegun(StackGamma stackgamma, int3 dimvol,
										 float posx, float posy, float posz,
										 float dx, float dy, float dz, float E, float rad) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (id < stackgamma.size) {
		float phi, r;
		int seed;
		if (stackgamma.in[id]==0 || stackgamma.live[id]==0) {
			seed = stackgamma.seed[id];
			phi = park_miller_jb(&seed) * twopi;
			r   = park_miller_jb(&seed) * rad;
			stackgamma.seed[id] = seed;
			stackgamma.E[id] = E;
			stackgamma.px[id] = posx + r * __cosf(phi);
			stackgamma.py[id] = posy;
			stackgamma.pz[id] = posz + r * __sinf(phi);
			stackgamma.dx[id] = dx;
			stackgamma.dy[id] = dy;
			stackgamma.dz[id] = dz;
			stackgamma.live[id] = 1;
			stackgamma.in[id] = 1;
			stackgamma.ct_eff[id] = 0;
			stackgamma.ct_Cpt[id] = 0;
			stackgamma.ct_PE[id] = 0;
		}
	}

}

/***********************************************************
 * Tracking kernel
 ***********************************************************/
__global__ void kernel_siddon(int3 dimvol, StackGamma stackgamma, float* dtrack, float dimvox) {

	int3 u, i, e, stepi;
	float3 p0, pe, stept, astart, run, delta;
	float pq, oldv, newv, totv, val, E;
	float eps = 1.0e-5f;
	unsigned int id = __umul24(blockIdx.x, blockDim.x)+threadIdx.x;
	int jump = dimvol.x*dimvol.y;
	int seed, inside, oldmat, mat;
	
	if (id < stackgamma.size) {
		p0.x = stackgamma.px[id];
		p0.y = stackgamma.py[id];
		p0.z = stackgamma.pz[id];
		delta.x = stackgamma.dx[id];
		delta.y = stackgamma.dy[id];
		delta.z = stackgamma.dz[id];
		seed = stackgamma.seed[id];
		E = stackgamma.E[id];

		// get free mean path
		//oldmat = dvol[int(p0.z)*jump + int(p0.y)*dimvol.x + int(p0.x)];
		oldmat = tex1Dfetch(tex_vol, int(p0.z)*jump + int(p0.y)*dimvol.x + int(p0.x));
		pq = -__fdividef(__logf(park_miller_jb(&seed)), att_from_mat(oldmat, E));
		pq = __fdividef(pq, dimvox);
		
		pe.x = p0.x + delta.x*pq;
		pe.y = p0.y + delta.y*pq;
		pe.z = p0.z + delta.z*pq;

		e.x = int(p0.x);
		e.y = int(p0.y);
		e.z = int(p0.z);

		if ((pe.x-p0.x) > 0) {stepi.x = 1; u.x = e.x + 1;}
		if ((pe.x-p0.x) < 0) {stepi.x = -1; u.x = e.x;}
		if ((pe.x-p0.x) == 0) {stepi.x = 0; u.x = e.x; pe.x = eps;}

		if ((pe.y-p0.y) > 0) {stepi.y = 1; u.y = e.y+1;}
		if ((pe.y-p0.y) < 0) {stepi.y = -1; u.y = e.y;}
		if ((pe.y-p0.y) == 0) {stepi.y = 0; u.y = e.y; pe.y = eps;}

		if ((pe.z-p0.z) > 0) {stepi.z = 1; u.z = e.z+1;}
		if ((pe.z-p0.z) < 0) {stepi.z = -1; u.z = e.z;}
		if ((pe.z-p0.z) == 0) {stepi.z = 0; u.z = e.z; pe.z = eps;}

		astart.x = (u.x - p0.x) / (pe.x - p0.x);
		astart.y = (u.y - p0.y) / (pe.y - p0.y);
		astart.z = (u.z - p0.z) / (pe.z - p0.z);

		run.x = astart.x * pq;
		run.y = astart.y * pq;
		run.z = astart.z * pq;
		oldv = run.x;
		if (run.y < oldv) {oldv = run.y;}
		if (run.z < oldv) {oldv = run.z;}

		stept.x = fabsf((pq / (pe.x - p0.x)));
		stept.y = fabsf((pq / (pe.y - p0.y)));
		stept.z = fabsf((pq / (pe.z - p0.z)));
		i.x = e.x;
		i.y = e.y;
		i.z = e.z;

		if (run.x == oldv) {run.x += stept.x; i.x += stepi.x;}
		if (run.y == oldv) {run.y += stept.y; i.y += stepi.y;}
		if (run.z == oldv) {run.z += stept.z; i.z += stepi.z;}

		// to debug
		//dtrack[e.z*jump + e.y*dimvol.x + e.x] += oldv;
		
		totv = 0.0f;
		inside = 1;
		while ((oldv < pq) & inside) {
			newv = run.x;
			if (run.y < newv) {newv=run.y;}
			if (run.z < newv) {newv=run.z;}
			val = (newv - oldv);

			// if mat change
			//mat = dvol[i.z*jump + i.y*dimvol.x + i.x];
			mat = tex1Dfetch(tex_vol, i.z*jump + i.y*dimvol.x + i.x);
			if (mat != oldmat) {
				pq = -__fdividef(__logf(park_miller_jb(&seed)), att_from_mat(oldmat, E));
				oldmat = mat;
			}

			//dtrack[i.z*jump + i.y*dimvol.x + i.x] += val;

			totv += val;
			oldv = newv;
			if (run.x==newv) {i.x += stepi.x; run.x += stept.x;}
			if (run.y==newv) {i.y += stepi.y; run.y += stept.y;}
			if (run.z==newv) {i.z += stepi.z; run.z += stept.z;}
			inside = (i.x >= 0) & (i.x < dimvol.x) & (i.y >= 0) & (i.y < dimvol.y) & (i.z >= 0) & (i.z < dimvol.z);
		}

		pe.x = p0.x + delta.x*oldv;
		pe.y = p0.y + delta.y*oldv;
		pe.z = p0.z + delta.z*oldv;
		stackgamma.seed[id] = seed;
		stackgamma.px[id] = pe.x;
		stackgamma.py[id] = pe.y;
		stackgamma.pz[id] = pe.z;

		if (!inside) {stackgamma.in[id] = 0;}

	} // id < nx

}

/*
__global__ void kernel_amanatides(float* dvol, float* dX0, float* dY0, float* dZ0,
								  float* dXe, float* dYe, float* dZe, int nx0, int jump, int nx) {

	int3 u, i, e, stepi;
	float3 p0, pe, stept, astart, run;
	float pq, oldv, totv, mu, val;
	float eps = 1.0e-5f;
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < nx0) {
		p0.x = dX0[id];
		p0.y = dY0[id];
		p0.z = dZ0[id];
		pe.x = dXe[id];
		pe.y = dYe[id];
		pe.z = dZe[id];

		e.x = int(p0.x);
		e.y = int(p0.y);
		e.z = int(p0.z);

		if ((pe.x-p0.x) > 0) {stepi.x = 1; u.x = e.x + 1;}
		if ((pe.x-p0.x) < 0) {stepi.x = -1; u.x = e.x;}
		if ((pe.x-p0.x) == 0) {stepi.x = 0; u.x = e.x; pe.x = eps;}

		if ((pe.y-p0.y) > 0) {stepi.y = 1; u.y = e.y+1;}
		if ((pe.y-p0.y) < 0) {stepi.y = -1; u.y = e.y;}
		if ((pe.y-p0.y) == 0) {stepi.y = 0; u.y = e.y; pe.y = eps;}

		if ((pe.z-p0.z) > 0) {stepi.z = 1; u.z = e.z+1;}
		if ((pe.z-p0.z) < 0) {stepi.z = -1; u.z = e.z;}
		if ((pe.z-p0.z) == 0) {stepi.z = 0; u.z = e.z; pe.z = eps;}

		astart.x = (u.x - p0.x) / (pe.x - p0.x);
		astart.y = (u.y - p0.y) / (pe.y - p0.y);
		astart.z = (u.z - p0.z) / (pe.z - p0.z);
		
		pq = sqrtf((p0.x-pe.x)*(p0.x-pe.x)+(p0.y-pe.y)*(p0.y-pe.y)+(p0.z-pe.z)*(p0.z-pe.z));
		run.x = astart.x * pq;
		run.y = astart.y * pq;
		run.z = astart.z * pq;
		oldv = run.x;
		if (run.y < oldv) {oldv = run.y;}
		if (run.z < oldv) {oldv = run.z;}

		stept.x = fabsf((pq / (pe.x - p0.x)));
		stept.y = fabsf((pq / (pe.y - p0.y)));
		stept.z = fabsf((pq / (pe.z - p0.z)));
		i.x = e.x;
		i.y = e.y;
		i.z = e.z;

		mu = oldv*dvol[e.z*jump + e.y*nx + e.x];
		//dvol[e.z*jump + e.y*nx + e.x] += oldv;
		
		totv = 0.0f;
		while (totv < pq) {
			if (run.x < run.y) {
				if (run.x < run.z) {i.x += stepi.x; run.x += stept.x;}
				else {i.z += stepi.z; run.z += stept.z;}
			} else {
				if (run.y < run.z) {i.y += stepi.y; run.y += stept.y;}
				else {i.z += stepi.z; run.z += stept.z;}
			}
			totv = run.x;
			if (run.y < totv) {totv=run.y;}
			if (run.z < totv) {totv=run.z;}
			val = totv-oldv;
			mu = val * dvol[i.z*jump + i.y*nx + i.x];
			//dvol[i.z*jump + i.y*nx + i.x] += val;
			oldv = totv;
		}
		
		mu = (pq-totv)*dvol[i.z*jump + i.y*nx + i.x];
		//dvol[i.z*jump + i.y*nx + i.x] += (pq - totv);

	} // id < nx

}
*/
/*
__global__ void kernel_raypro(float* dvol, int3 dimvol, StackGamma stackgamma) {
	float3 xi, x0, d, rd, db, sd;
	int3 p, b, ob;
	float t, tn, tot_t, dist, mu, phi, theta;
	float eps = 1.0e-5f;
	int jump = dimvol.x*dimvol.y;

	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int seed = stackgamma.seed[id];
	int inside;
	int watchdog;
	if (id < stackgamma.size) {
		x0.x = stackgamma.px[id];
		x0.y = stackgamma.py[id];
		x0.z = stackgamma.pz[id];
		d.x = stackgamma.dx[id];
		d.y = stackgamma.dy[id];
		d.z = stackgamma.dz[id];

		dist = -__logf(park_miller_jb(&seed)) / 0.018f;

		if (d.x==0) {d.x=eps;}
		if (d.y==0) {d.y=eps;}
		if (d.z==0) {d.z=eps;}

		rd.x = __fdividef(1.0f, d.x);
		rd.y = __fdividef(1.0f, d.y);
		rd.z = __fdividef(1.0f, d.z);

		db.x = (d.x > 0) - (d.x < 0) * eps;
		db.y = (d.y > 0) - (d.y < 0) * eps;
		db.z = (d.z > 0) - (d.z < 0) * eps;

		b.x = int(x0.x+db.x);
		b.y = int(x0.y+db.y);
		b.z = int(x0.z+db.z);
		ob.x = b.x; ob.y = b.y; ob.z = b.z;

		t = (b.x - x0.x) * rd.x;
		tn = (b.y - x0.y) * rd.y;
		t = fminf(t, tn);
		tn = (b.z - x0.z) * rd.z;
		t = fminf(t, tn);

		xi.x = x0.x + (d.x * t);
		xi.y = x0.y + (d.y * t);
		xi.z = x0.z + (d.z * t);

		tn = 1.0f + int(xi.x) - xi.x;
		xi.x += (tn * (tn < eps));
		tn = 1.0f + int(xi.y) - xi.y;
		xi.y += (tn * (tn < eps));
		tn = 1.0f + int(xi.z) - xi.z;
		xi.z += (tn * (tn < eps));

		tot_t = t;
		p.x = int(x0.x);
		p.y = int(x0.y);
		p.z = int(x0.z);

		inside = 1;
		watchdog=0;
		while ((tot_t < dist) & inside) {
			mu = t * dvol[p.z*jump + p.y*dimvol.x + p.x];
			//dvol[p.z*jump + p.y*dimvol.x + p.x] += t;
			
			b.x = int(xi.x + db.x);
			b.y = int(xi.y + db.y);
			b.z = int(xi.z + db.z);

			t = (b.x - xi.x) * rd.x;
			tn = (b.y - xi.y) * rd.y;
			t = fminf(t, tn);
			tn = (b.z - xi.z) * rd.z;
			t = fminf(t, tn);
			
			xi.x = xi.x + (d.x * t);
			xi.y = xi.y + (d.y * t);
			xi.z = xi.z + (d.z * t);

			tot_t += t;
			p.x += (b.x - ob.x);
			p.y += (b.y - ob.y);
			p.z += (b.z - ob.z);
			ob.x = b.x; ob.y = b.y; ob.z = b.z;
			
			inside = (p.x >= 0) & (p.x < dimvol.x) & (p.y >= 0) & (p.y < dimvol.y) & (p.z >= 0) & (p.z < dimvol.z);
			dvol[watchdog] = p.x;
			++watchdog;
			if (watchdog > 500) {
				//dvol[0] = b.x;
				//dvol[1] = p.z;
				break;
			}

		}

		if (!inside) {
			stackgamma.in[id] = 0;
			return;
		}

		mu = (dist-tot_t) * dvol[p.z*jump + p.y*dimvol.x + p.x];
		//dvol[p.z*jump + p.y*dimvol.x + p.x] += (dist-tot_t);

		stackgamma.seed[id] = seed;
		
	} // id
	
}
*/

/***********************************************************
 * Main
 ***********************************************************/
void mc_cuda(float* vol, int nz, int ny, int nx,
			 float* E, int nE, float* dx, int ndx, float* dy, int ndy, float* dz, int ndz,
			 float* px, int npx, float* py, int npy, float* pz, int npz,
			 int nparticles) {
	hipSetDevice(1);

    timeval start, end;
    double t1, t2, diff;
	int3 dimvol;
	int n, step;
	int countparticle=0;
	
	dimvol.x = nx;
	dimvol.y = ny;
	dimvol.z = nz;

	// Volume allocation
	unsigned int mem_vol = nz*ny*nx * sizeof(float);
	float* dvol;
	hipMalloc((void**) &dvol, mem_vol);
	hipMemcpy(dvol, vol, mem_vol, hipMemcpyHostToDevice);
	hipBindTexture(NULL, tex_vol, dvol, mem_vol);
	float* dtrack;
	hipMalloc((void**) &dtrack, mem_vol);
	hipMemset(dtrack, 0, mem_vol);
	float* ddose;
	hipMalloc((void**) &ddose, mem_vol);
	hipMemset(ddose, 0, mem_vol);

	// Stacks
	StackGamma stackgamma;
	StackGamma collector;
	stackgamma.size = nparticles;
	//unsigned int mem_stack = stackgamma.size * sizeof(stackgamma);
	unsigned int mem_stack_float = stackgamma.size * sizeof(float);
	unsigned int mem_stack_int = stackgamma.size * sizeof(int);
	unsigned int mem_stack_char = stackgamma.size * sizeof(char);

	// Host stack allocation memory
	collector.E = (float*)malloc(mem_stack_float);
	collector.dx = (float*)malloc(mem_stack_float);
	collector.dy = (float*)malloc(mem_stack_float);
	collector.dz = (float*)malloc(mem_stack_float);
	collector.px = (float*)malloc(mem_stack_float);
	collector.py = (float*)malloc(mem_stack_float);
	collector.pz = (float*)malloc(mem_stack_float);
	collector.live = (unsigned char*)malloc(mem_stack_char);
	collector.in = (unsigned char*)malloc(mem_stack_char);
	collector.ct_eff = (int*)malloc(mem_stack_int);
	collector.ct_Cpt = (int*)malloc(mem_stack_int);
	collector.ct_PE = (int*)malloc(mem_stack_int);

	// Device stack allocation memory
	hipMalloc((void**) &stackgamma.E, mem_stack_float);
	hipMalloc((void**) &stackgamma.dx, mem_stack_float);
	hipMalloc((void**) &stackgamma.dy, mem_stack_float);
	hipMalloc((void**) &stackgamma.dz, mem_stack_float);
	hipMalloc((void**) &stackgamma.px, mem_stack_float);
	hipMalloc((void**) &stackgamma.py, mem_stack_float);
	hipMalloc((void**) &stackgamma.pz, mem_stack_float);
	hipMalloc((void**) &stackgamma.seed, mem_stack_int);
	hipMalloc((void**) &stackgamma.live, mem_stack_char);
	hipMalloc((void**) &stackgamma.in, mem_stack_char);
	hipMalloc((void**) &stackgamma.ct_eff, mem_stack_int);
	hipMalloc((void**) &stackgamma.ct_Cpt, mem_stack_int);
	hipMalloc((void**) &stackgamma.ct_PE, mem_stack_int);
	hipMemset(stackgamma.live, 0, mem_stack_char); // at beginning all particles are dead
	hipMemset(stackgamma.in, 0, mem_stack_char);   // and outside the volume
	
	// Init seeds
	int* tmp = (int*)malloc(stackgamma.size * sizeof(int));
	srand(10);
	n=0;
	while (n<stackgamma.size) {tmp[n] = rand(); ++n;}
	hipMemcpy(stackgamma.seed, tmp, mem_stack_int, hipMemcpyHostToDevice);
	free(tmp);

	// Vars kernel
	dim3 threads, grid;
	int block_size = 256;
	int grid_size = (nparticles + block_size - 1) / block_size;
	threads.x = block_size;
	grid.x = grid_size;

	// Outter loop
	for (step=0; step<2; ++step) {
		printf("Step %i\n", step);
		// Init particles
		gettimeofday(&start, NULL);
		t1 = start.tv_sec + start.tv_usec / 1000000.0;
		kernel_particle_largegun<<<grid, threads>>>(stackgamma, dimvol, 45.0, 0.0, 35.0, 0.0, 1.0, 0.0, 0.511, 5.0);
		hipDeviceSynchronize();
		gettimeofday(&end, NULL);
		t2 = end.tv_sec + end.tv_usec / 1000000.0;
		diff = t2 - t1;
		printf("   Create gamma particles %f s\n", diff);
	
		// Propagation
		gettimeofday(&start, NULL);
		t1 = start.tv_sec + start.tv_usec / 1000000.0;
		kernel_siddon<<<grid, threads>>>(dimvol, stackgamma, dtrack, 4.0); // 4.0 mm3 voxel
		hipDeviceSynchronize();
		gettimeofday(&end, NULL);
		t2 = end.tv_sec + end.tv_usec / 1000000.0;
		diff = t2 - t1;
		printf("   Track gamma particles %f s\n", diff);

		// Interactions
		gettimeofday(&start, NULL);
		t1 = start.tv_sec + start.tv_usec / 1000000.0;
		kernel_interactions<<<grid, threads>>>(stackgamma, ddose, dimvol);
		hipDeviceSynchronize();
		gettimeofday(&end, NULL);
		t2 = end.tv_sec + end.tv_usec / 1000000.0;
		diff = t2 - t1;
		printf("   Interactions gamma particles %f s\n", diff);

		// Collector
		gettimeofday(&start, NULL);
		t1 = start.tv_sec + start.tv_usec / 1000000.0;
		hipMemcpy(collector.E, stackgamma.E, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.dx, stackgamma.dx, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.dy, stackgamma.dy, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.dz, stackgamma.dz, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.px, stackgamma.px, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.py, stackgamma.py, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.pz, stackgamma.pz, mem_stack_float, hipMemcpyDeviceToHost);
		hipMemcpy(collector.live, stackgamma.live, mem_stack_char, hipMemcpyDeviceToHost);
		hipMemcpy(collector.in, stackgamma.in, mem_stack_char, hipMemcpyDeviceToHost);
		hipMemcpy(collector.ct_eff, stackgamma.ct_eff, mem_stack_int, hipMemcpyDeviceToHost);
		hipMemcpy(collector.ct_Cpt, stackgamma.ct_Cpt, mem_stack_int, hipMemcpyDeviceToHost);
		hipMemcpy(collector.ct_PE, stackgamma.ct_PE, mem_stack_int, hipMemcpyDeviceToHost);			
		gettimeofday(&end, NULL);
		t2 = end.tv_sec + end.tv_usec / 1000000.0;
		diff = t2 - t1;
		printf("   Get back stack of gamma particles %f s\n", diff);

		gettimeofday(&start, NULL);
		t1 = start.tv_sec + start.tv_usec / 1000000.0;
		int c1=0;
		int c2=0;
		int c3=0;
		int c4=0;
		n=0;
		while(n<nparticles && countparticle<nparticles) {
			if (collector.in[n] == 0) {
				E[countparticle] = collector.E[n];
				dx[countparticle] = collector.dx[n];
				dy[countparticle] = collector.dy[n];
				dz[countparticle] = collector.dz[n];
				px[countparticle] = collector.px[n];
				py[countparticle] = collector.py[n];
				pz[countparticle] = collector.pz[n];
				++countparticle;
			}
			if (collector.live[n] == 0) {++c1;}
			c2 += collector.ct_eff[n];
			c3 += collector.ct_Cpt[n];
			c4 += collector.ct_PE[n];
			++n;
		}
		gettimeofday(&end, NULL);
		t2 = end.tv_sec + end.tv_usec / 1000000.0;
		diff = t2 - t1;
		
		printf("   Store gamma particles %f s\n", diff);
		printf("   Nb particles outside %i absorbed %i\n", countparticle, c1);
		printf("   Tot interaction %i: %i Compton %i Photo-Electric\n", c2, c3, c4);

	} // outter loop (step)
	
	//hipMemcpy(tmp, stackgamma.seed, mem_stack_int, hipMemcpyDeviceToHost);
	//hipMemcpy(vol, dtrack, mem_vol, hipMemcpyDeviceToHost);
	hipMemcpy(vol, ddose, mem_vol, hipMemcpyDeviceToHost);

	// Clean memory
	free(collector.E);
	free(collector.dx);
	free(collector.dy);
	free(collector.dz);
	free(collector.px);
	free(collector.py);
	free(collector.pz);
	free(collector.live);
	free(collector.in);
	free(collector.ct_eff);
	free(collector.ct_Cpt);
	free(collector.ct_PE);
	
	hipUnbindTexture(tex_vol);
	hipFree(dvol);
	hipFree(ddose);
	hipFree(stackgamma.E);
	hipFree(stackgamma.dx);
	hipFree(stackgamma.dy);
	hipFree(stackgamma.dz);
	hipFree(stackgamma.px);
	hipFree(stackgamma.py);
	hipFree(stackgamma.pz);
	hipFree(stackgamma.live);
	hipFree(stackgamma.in);
	hipFree(stackgamma.seed);
	hipFree(stackgamma.ct_eff);
	hipFree(stackgamma.ct_Cpt);
	hipFree(stackgamma.ct_PE);
	hipDeviceReset();

}
